#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include "sceneStructs.h"

static int blockSize = 1024;
static dim3 blockNum;

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
//Non-optimized Scan:
		__global__ void non_opt_cudaSweepUp(int n, int d, int *data) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			int interval_length = 1 << (d + 1);
			if (index >= n)
				return;
			if (index % interval_length == 0) {
				data[index + (1 << (d + 1)) - 1] += data[index + (1 << d) - 1];
			}
		}

		__global__ void non_opt_cudaSweepDown(int n, int d, int *data) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			int interval_length = 1 << (d + 1);
			// k from 0 to n-1
			if (index >= n)
				return;
			if (index % interval_length == 0) {
				int temp = data[index + (1 << d) - 1];
				data[index + (1 << d) - 1] = data[index + (1 << (d + 1)) - 1];
				data[index + (1 << (d + 1)) - 1] += temp;
			}
		}

		void non_opt_scan(int n, int *odata, const int *idata) {
			// TODO
			if (n <= 0)
				return;
			int celllog = ilog2ceil(n);

			int pow2len = 1 << celllog;

			int *dev_data;
			hipMalloc((void**)&dev_data, pow2len * sizeof(int));
			checkCUDAError("hipMalloc dev_data failed!");

			hipMemcpy(dev_data, idata, pow2len * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy failed!");

			timer().startGpuTimer();

			//Up-Sweep
			for (int d = 0; d <= celllog - 1; d++) {
				blockNum = (pow2len + blockSize) / blockSize;
				non_opt_cudaSweepUp << <blockNum, blockSize >> >(pow2len, d, dev_data);
			}

			//hipMemcpy(odata, dev_data, n * sizeof(int), hipMemcpyDeviceToHost);

			//Down-Sweep
			hipMemset(dev_data + pow2len - 1, 0, sizeof(int));
			checkCUDAError("hipMemset failed!");

			for (int d = celllog - 1; d >= 0; d--) {
				blockNum = (pow2len + blockSize) / blockSize;
				non_opt_cudaSweepDown << <blockNum, blockSize >> >(pow2len, d, dev_data);
			}
			timer().endGpuTimer();

			hipMemcpy(odata, dev_data, n * sizeof(int), hipMemcpyDeviceToHost);

			checkCUDAError("hipMalloc dev_data to odata failed!");

			hipFree(dev_data);
			checkCUDAError("hipFree dev_data failed!");

		}

//Optimized Scan
		__global__ void cudaSweepUp(int n, int d, int *data) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);		
			int interval_length = 1 << (d + 1);
			if (index >= n)
				return;
			//int idx1 = index * interval_length + (1 << (d + 1)) - 1;
			//int idx2 = index * interval_length + (1 << d) - 1;
			data[index * interval_length + (1 << (d + 1)) - 1] += data[index * interval_length + (1 << d) - 1];
		}

		__global__ void cudaSweepDown(int n, int d, int *data) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			int interval_length = 1 << (d + 1);
			// k from 0 to n-1
			if (index >= n)
				return;

			int temp = data[index * interval_length + (1 << d) - 1];
			data[index * interval_length + (1 << d) - 1] = data[index * interval_length + (1 << (d + 1)) - 1];
			data[index * interval_length + (1 << (d + 1)) - 1] += temp;
		}

        void scan(int n, int *odata, const int *idata) {
            // TODO
			if (n <= 0)
				return;
			int celllog = ilog2ceil(n);

			int pow2len = 1 << celllog;

			int *dev_data;
			
			hipMalloc((void**)&dev_data, pow2len * sizeof(int));
			checkCUDAError("hipMalloc dev_data failed!");

			hipMemcpy(dev_data, idata, pow2len * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy failed!");
			
			timer().startGpuTimer();

			//Up-Sweep
			for (int d = 0; d <= celllog - 1; d++) {
				int interval_length = (1 << (d + 1));
				blockNum = (pow2len / interval_length + blockSize) / blockSize;
				cudaSweepUp<<<blockNum, blockSize>>>(pow2len / interval_length, d, dev_data);
			}
			
			//hipMemcpy(odata, dev_data, n * sizeof(int), hipMemcpyDeviceToHost);

			//Down-Sweep
			hipMemset(dev_data + pow2len - 1, 0, sizeof(int));
			checkCUDAError("hipMemset failed!");

			for (int d = celllog - 1; d >= 0; d--) {
				int num_operations = (1 << (d + 1));
				blockNum = (pow2len / num_operations + blockSize) / blockSize;
				cudaSweepDown<<<blockNum, blockSize >>>(pow2len / num_operations, d, dev_data);
			}
			timer().endGpuTimer();

			hipMemcpy(odata, dev_data, n * sizeof(int), hipMemcpyDeviceToHost);

			checkCUDAError("cudaMmcpy dev_data to odata failed!");

			hipFree(dev_data);
			checkCUDAError("hipFree dev_data failed!");

        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            // TODO
			if (n <= 0)
				return -1;
			int celllog = ilog2ceil(n);
			int pow2len = 1 << celllog;

			int *dev_idata, *dev_odata, *dev_bool_data, *dev_indices;
			hipMalloc((void**)&dev_idata, pow2len * sizeof(int));
			checkCUDAError("hipMalloc dev_idata failed!");
			hipMalloc((void**)&dev_odata, pow2len * sizeof(int));
			checkCUDAError("hipMalloc dev_odata failed!");
			hipMalloc((void**)&dev_bool_data, pow2len * sizeof(int));
			checkCUDAError("hipMalloc dev_bool_data failed!");
			hipMalloc((void**)&dev_indices, pow2len * sizeof(int));
			checkCUDAError("hipMalloc dev_indices failed!");
			
		//bool Mapping
			hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
			timer().startGpuTimer();
			blockNum = (n + blockSize) / blockSize;
			Common::kernMapToBoolean<<<blockNum, blockSize>>>(n, dev_bool_data, dev_idata);
		// Scan
			hipMemcpy(dev_indices, dev_bool_data, pow2len * sizeof(int), hipMemcpyDeviceToDevice);
			checkCUDAError("hipMemcpy failed!");
			dev_bool_data;

			//Up-Sweep
			for (int d = 0; d <= celllog - 1; d++) {
				int interval_length = (1 << (d + 1));
				blockNum = (pow2len / interval_length + blockSize) / blockSize;
				cudaSweepUp << <blockNum, blockSize >> >(pow2len / interval_length, d, dev_indices);
			}


			//Down-Sweep
			hipMemset(dev_indices + pow2len - 1, 0, sizeof(int));
			checkCUDAError("hipMemset failed!");

			for (int d = celllog - 1; d >= 0; d--) {
				int num_operations = (1 << (d + 1));
				blockNum = (pow2len / num_operations + blockSize) / blockSize;
				cudaSweepDown << <blockNum, blockSize >> >(pow2len / num_operations, d, dev_indices);
			}

			
		//Scattered
			blockNum = (n + blockSize) / blockSize;
			Common::kernScatter<<<blockNum, blockSize>>>(n, dev_odata, dev_idata, dev_bool_data, dev_indices);
			
			timer().endGpuTimer();
			//compute count
			int a, b;
			hipMemcpy(&a, dev_bool_data + (n - 1), sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(&b, dev_indices + (n - 1), sizeof(int), hipMemcpyDeviceToHost);
			int count = a + b;
			hipMemcpy(odata, dev_odata, count * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy dev_odata to odata failed!");

		//Free data
			hipFree(dev_idata);
			checkCUDAError("hipFree dev_idata failed!");
			hipFree(dev_odata);
			checkCUDAError("hipFree dev_idata failed!");
			hipFree(dev_bool_data);
			checkCUDAError("hipFree dev_idata failed!");
			hipFree(dev_indices);
			checkCUDAError("hipFree dev_idata failed!");


            return count;
        }
    }
}
