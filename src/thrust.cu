#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
			thrust::device_vector<int> dev_idata(idata, idata + n);
			thrust::device_vector<int> dev_odata(odata, odata + n);
            timer().startGpuTimer();
			thrust::exclusive_scan(dev_idata.begin(), dev_idata.end(), dev_odata.begin());
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
            timer().endGpuTimer();
			thrust::copy(dev_odata.begin(), dev_odata.end(), odata);
        }
    }
}
