#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtc/matrix_inverse.hpp>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1

#define CACHEFIRST 0
#define TRACERENDER 0
#define REALISTICCAMERA 0
#define SORTBYMATERIAL 0

#define NEAR_CLIP 0.1f
#define FAR_CLIP 100.0f
#define focalDistance 15.5f
#define lensRadius 20.0f

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (hipSuccess == err) {
		return;
	}

	fprintf(stderr, "CUDA error");
	if (file) {
		fprintf(stderr, " (%s:%d)", file, line);
	}
	fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
	getchar();
#  endif
	exit(EXIT_FAILURE);
#endif
}

struct DeadPath {
	__host__ __device__
		bool operator()(const PathSegment &path) {
		return path.remainingBounces <= 0;
	}
};

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
	int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
	int iter, glm::vec3* image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);
		glm::vec3 pix = image[index];

		glm::ivec3 color;
		color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
		color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
		color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = color.x;
		pbo[index].y = color.y;
		pbo[index].z = color.z;
	}
}

static Scene * hst_scene = NULL;
static glm::vec3 * dev_image = NULL;
static Geom * dev_geoms = NULL;
static Material * dev_materials = NULL;
static PathSegment * dev_paths = NULL;
static ShadeableIntersection * dev_intersections = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...
int *dev_mId = NULL;
static ShadeableIntersection * dev_first_intersections = NULL;

void pathtraceInit(Scene *scene) {
	hst_scene = scene;
	//scene->state.traceDepth = 8;
	const Camera &cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	// TODO: initialize any extra device memeory you need
	hipMalloc((void**)&dev_mId, pixelcount * sizeof(int));
	hipMalloc((void**)&dev_first_intersections, pixelcount * sizeof(ShadeableIntersection));

	checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
	hipFree(dev_image);  // no-op if dev_image is null
	hipFree(dev_paths);
	hipFree(dev_geoms);
	hipFree(dev_materials);
	hipFree(dev_intersections);
	// TODO: clean up any extra device memory you created
	hipFree(dev_mId);
	hipFree(dev_first_intersections);

	checkCUDAError("pathtraceFree");
}

__device__ glm::mat4 Rebuild(glm::vec3 translation, glm::vec3 rotation, glm::vec3 scale) {
	glm::mat4 translationMat = glm::translate(glm::mat4(), translation);
	glm::mat4 rotationMat = glm::rotate(glm::mat4(), rotation.x * (float)PI / 180, glm::vec3(1, 0, 0));
	rotationMat = rotationMat * glm::rotate(glm::mat4(), rotation.y * (float)PI / 180, glm::vec3(0, 1, 0));
	rotationMat = rotationMat * glm::rotate(glm::mat4(), rotation.z * (float)PI / 180, glm::vec3(0, 0, 1));
	glm::mat4 scaleMat = glm::scale(glm::mat4(), scale);
	return translationMat * rotationMat * scaleMat;
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__device__ glm::vec3 squareToDiskConcentric(const glm::vec2 &sample)
{
	float r, angle;
	float a = 2 * sample[0] - 1;
	float b = 2 * sample[1] - 1;

	if (a > ((-1)*b))
	{
		if (a > b)
		{
			r = a;
			angle = (PI / 4) * (b / a);
		}
		else
		{
			r = b;
			angle = (PI / 4) * (2.0f - (a / b));
		}
	}
	else
	{
		if (a < b)
		{
			r = (-1) * a;
			angle = (PI / 4) * (4 + b / a);
		}
		else
		{
			r = (-1) * b;
			if (fabs(b) > 0.00001f )
				angle = (PI / 4) * (6.0f - (a / b));
			else
				angle = 0;
		}
	}

	glm::vec3 result(r * cos(angle), r * sin(angle), 0);
	return result;
}

__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments, Geom *dev_geoms)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment & segment = pathSegments[index];

		segment.ray.origin = cam.position;
		segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

		// TODO: implement antialiasing by jittering the ray
		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
		);
#if REALISTICCAMERA
		float ndc_x = 2 * float(x) / cam.resolution.x - 1.0f;
		ndc_x *= -1;
		float ndc_y = 1.0f - 2 * float(y) / cam.resolution.y;
		glm::vec3 ndc_point = glm::vec3(ndc_x, ndc_y, 0.f);
		glm::mat4 presp_mat = glm::perspective(glm::radians(cam.fov[1]), cam.resolution.x / (float)cam.resolution.y, NEAR_CLIP, FAR_CLIP);
		glm::mat4 inv_persp_mat = glm::inverse(presp_mat);
		glm::vec4 camera_point_vec4 = inv_persp_mat * glm::vec4(ndc_point, 1.f);
		glm::vec3 camera_point = glm::vec3(camera_point_vec4);
		Ray r;
		r.origin = glm::vec3(0.f, 0.f, 0.f);
		r.direction = glm::normalize(camera_point);
		float IOR = 1.5f;
		float templensRadius = focalDistance * (IOR - 1.f);
		if (lensRadius > 0)
		{
			//Sample point on lens
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
			thrust::uniform_real_distribution<float> u01(0, 1);
			glm::vec2 pLens = glm::vec2(templensRadius * squareToDiskConcentric(glm::vec2(u01(rng), u01(rng))));
			//Compute point on plane of focus
			float ft = focalDistance / fabs(r.direction[2]);
			glm::vec3 pFocus = r.origin + r.direction * ft;
			//Update ray for effect of lens 375
			r.origin = glm::vec3(pLens.x, pLens.y, 0);
			r.direction = glm::normalize(pFocus - r.origin);
		}
		glm::mat4 cameraToWorld = glm::lookAt(cam.position, cam.lookAt, cam.up);
		r.origin = glm::vec3(glm::inverse(cameraToWorld) * glm::vec4(r.origin, 1.f));
		r.direction = glm::normalize(glm::vec3(glm::vec3(glm::inverse(cameraToWorld) * glm::vec4(r.direction, 0.f))));
		segment.ray = r;
#endif

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
#if TRACERENDER
		if (index == 0)
		{
			dev_geoms[6].translation += glm::vec3(0.02f, 0.02f, 0.00f);
			dev_geoms[6].transform = Rebuild(dev_geoms[6].translation, dev_geoms[6].rotation, dev_geoms[6].scale);
			dev_geoms[6].inverseTransform = glm::inverse(dev_geoms[6].transform);
			dev_geoms[6].invTranspose = glm::transpose(dev_geoms[6].inverseTransform);
		}
#endif	
	}
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment * pathSegments
	, Geom * geoms
	, int geoms_size
	, ShadeableIntersection * intersections
)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom & geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			// TODO: add more intersection tests here... triangle? metaball? CSG?

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;
		}
	}
}

__global__ void setMaterialId(int *mId, int num_paths, ShadeableIntersection * intersections)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		mId[idx] = intersections[idx].materialId;
	}
}


// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial(
	int iter
	, int num_paths
	, ShadeableIntersection * shadeableIntersections
	, PathSegment * pathSegments
	, Material * materials
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) { // if the intersection exists...
									 // Set up the RNG
									 // LOOK: this is how you use thrust's RNG! Please look at
									 // makeSeededRandomEngine as well.
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
			thrust::uniform_real_distribution<float> u01(0, 1);

			Material material = materials[intersection.materialId];
			glm::vec3 materialColor = material.color;

			// If the material indicates that the object was a light, "light" the ray
			if (material.emittance > 0.0f) {
				pathSegments[idx].color *= (materialColor * material.emittance);
			}
			// Otherwise, do some pseudo-lighting computation. This is actually more
			// like what you would expect from shading in a rasterizer like OpenGL.
			// TODO: replace this! you should be able to start with basically a one-liner
			else {
				float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(0.0f, 1.0f, 0.0f));
				pathSegments[idx].color *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
				pathSegments[idx].color *= u01(rng); // apply some noise because why not
			}
			// If there was no intersection, color the ray black.
			// Lots of renderers use 4 channel color, RGBA, where A = alpha, often
			// used for opacity, in which case they can indicate "no opacity".
			// This can be useful for post-processing and image compositing.
		}
		else {
			pathSegments[idx].color = glm::vec3(0.0f);
		}
	}
}

__global__ void myShadeFakeMaterial(
	int iter
	, int num_paths
	, ShadeableIntersection * shadeableIntersections
	, PathSegment * pathSegments
	, Material * materials
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) { // if the intersection exists...
									 // Set up the RNG
									 // LOOK: this is how you use thrust's RNG! Please look at
									 // makeSeededRandomEngine as well.
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
			thrust::uniform_real_distribution<float> u01(0, 1);

			Material material = materials[intersection.materialId];
			glm::vec3 materialColor = material.color;

			// If the material indicates that the object was a light, "light" the ray
			if (material.emittance > 0.0f) {
				pathSegments[idx].color *= (materialColor * material.emittance);
				pathSegments[idx].remainingBounces = 0;
			}
			// Otherwise, do some pseudo-lighting computation. This is actually more
			// like what you would expect from shading in a rasterizer like OpenGL.
			// TODO: replace this! you should be able to start with basically a one-liner
			else {
				pathSegments[idx].color *= materialColor;
				glm::vec3 intersect = pathSegments[idx].ray.origin + intersection.t * glm::normalize(pathSegments[idx].ray.direction);
				pathSegments[idx].remainingBounces--;
				//if(pathSegments[idx].remainingBounces > 0)
					scatterRay(pathSegments[idx], intersect, intersection.surfaceNormal, material, rng);
				
			}
			// If there was no intersection, color the ray black.
			// Lots of renderers use 4 channel color, RGBA, where A = alpha, often
			// used for opacity, in which case they can indicate "no opacity".
			// This can be useful for post-processing and image compositing.
		}
		else {
			pathSegments[idx].color = glm::vec3(0.0f);
			pathSegments[idx].remainingBounces = 0;
		}
	}
}

__global__ void checkRemainBounces(int num_paths, PathSegment * pathSegments, glm::vec3 *image)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		if (pathSegments[idx].remainingBounces <= 0) {
			image[pathSegments[idx].pixelIndex] += pathSegments[idx].color;
			//pathSegments[idx].color = glm::vec3(0.0f);
		}
	}
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 * image, PathSegment * iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

/**
* Wrapper for the __global__ call that sets up the kernel calls and does a ton
* of memory management
*/
void pathtrace(uchar4 *pbo, int frame, int iter) {
	const int traceDepth = hst_scene->state.traceDepth;
	const Camera &cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

	///////////////////////////////////////////////////////////////////////////

	// Recap:
	// * Initialize array of path rays (using rays that come out of the camera)
	//   * You can pass the Camera object to that kernel.
	//   * Each path ray must carry at minimum a (ray, color) pair,
	//   * where color starts as the multiplicative identity, white = (1, 1, 1).
	//   * This has already been done for you.
	// * For each depth:
	//   * Compute an intersection in the scene for each path ray.
	//     A very naive version of this has been implemented for you, but feel
	//     free to add more primitives and/or a better algorithm.
	//     Currently, intersection distance is recorded as a parametric distance,
	//     t, or a "distance along the ray." t = -1.0 indicates no intersection.
	//     * Color is attenuated (multiplied) by reflections off of any object
	//   * TODO: Stream compact away all of the terminated paths.
	//     You may use either your implementation or `thrust::remove_if` or its
	//     cousins.
	//     * Note that you can't really use a 2D kernel launch any more - switch
	//       to 1D.
	//   * TODO: Shade the rays that intersected something or didn't bottom out.
	//     That is, color the ray by performing a color computation according
	//     to the shader, then generate a new ray to continue the ray path.
	//     We recommend just updating the ray's PathSegment in place.
	//     Note that this step may come before or after stream compaction,
	//     since some shaders you write may also cause a path to terminate.
	// * Finally, add this iteration's results to the image. This has been done
	//   for you.

	// TODO: perform one iteration of path tracing

	generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> >(cam, iter, traceDepth, dev_paths, dev_geoms);
	checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

	bool iterationComplete = false;

	//-----cuda event for testing runtime-----
	//  1.create and record
	/*
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	*/
	//----------------------------------------

	while (!iterationComplete) {
		
		// clean shading chunks
		hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

		// tracing
		dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
#if CACHEFIRST
		if (iter != 1 && depth == 0)
		{
			hipMemcpy(dev_intersections, dev_first_intersections, pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
		}
		else
		{ 
#endif
			computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
				depth
				, num_paths
				, dev_paths
				, dev_geoms
				, hst_scene->geoms.size()
				, dev_intersections
				);
#if CACHEFIRST
		}
		if (iter == 1 && depth == 0)
		{
			hipMemcpy(dev_first_intersections, dev_intersections, pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
		}
#endif
		checkCUDAError("trace one bounce");
		hipDeviceSynchronize();
		depth++;


#if SORTBYMATERIAL
		setMaterialId << <numblocksPathSegmentTracing, blockSize1d >> >(dev_mId, num_paths, dev_intersections);

		thrust::device_ptr<int> dev_thrust_Key(dev_mId);
		thrust::device_ptr<PathSegment> dev_thrust_paths(dev_paths);
		thrust::device_ptr<ShadeableIntersection> dev_thrust_intersections(dev_intersections);

		thrust::sort_by_key(dev_thrust_Key, dev_thrust_Key + num_paths, dev_thrust_paths);

		setMaterialId << <numblocksPathSegmentTracing, blockSize1d >> >(dev_mId, num_paths, dev_intersections);
		thrust::sort_by_key(dev_thrust_Key, dev_thrust_Key + num_paths, dev_thrust_intersections);
#endif
		// TODO:
		// --- Shading Stage ---
		// Shade path segments based on intersections and generate new rays by
		// evaluating the BSDF.
		// Start off with just a big kernel that handles all the different
		// materials you have in the scenefile.
		// TODO: compare between directly shading the path segments and shading
		// path segments that have been reshuffled to be contiguous in memory.

		myShadeFakeMaterial << <numblocksPathSegmentTracing, blockSize1d >> > (
			iter,
			num_paths,
			dev_intersections,
			dev_paths,
			dev_materials
			);
		
		checkRemainBounces << <numblocksPathSegmentTracing, blockSize1d >> > (num_paths, dev_paths, dev_image);

		//remove paths that remainingBounces <= 0, using thrust::remove_if
		PathSegment *dev_path_end = thrust::remove_if(thrust::device, dev_paths, dev_paths + num_paths, DeadPath());

		//Stop iteration when (reach max depth || no paths left)
		//Update num of paths.
		if (depth >= traceDepth || dev_path_end == dev_paths) {
			iterationComplete = true;
		}
		else {
			num_paths = dev_path_end - dev_paths;
		}

		
	}
	//-----cuda event for testing runtime-----
	//  2.record and compute
	/*
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float ms;
	hipEventElapsedTime(&ms, start, stop);
	if (iter == 3)
		cout << "run time of the whole iteration: " << ms << endl;
	*/
	//----------------------------------------


	// Assemble this iteration and apply it to the image
	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather << <numBlocksPixels, blockSize1d >> >(num_paths, dev_image, dev_paths);

	///////////////////////////////////////////////////////////////////////////

	// Send results to OpenGL buffer for rendering
	sendImageToPBO << <blocksPerGrid2d, blockSize2d >> >(pbo, cam.resolution, iter, dev_image);

	// Retrieve image from GPU
	hipMemcpy(hst_scene->state.image.data(), dev_image,
		pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	checkCUDAError("pathtrace");
}
