#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <stream_compaction/efficient.h>
#include <thrust/device_vector.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"


#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
    getchar();
#  endif
    exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
        int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene * hst_scene = NULL;
static glm::vec3 * dev_image = NULL;
static Geom * dev_geoms = NULL;
static Material * dev_materials = NULL;
static PathSegment * dev_paths = NULL;
static ShadeableIntersection * dev_intersections = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...
static int* dev_bool = NULL;//bool array for ray stream compaction
static int* dev_indices = NULL;//indices array for ray stream compaction
static PathSegment* dev_paths_helper = NULL;
static PathSegment* dev_paths_finished = NULL;

void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

  	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

  	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
  	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

  	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
  	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

  	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
  	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    // TODO: initialize any extra device memeory you need
	hipMalloc(&dev_indices, pow(2, ilog2ceil(pixelcount)) * sizeof(int));
	hipMalloc(&dev_bool, pow(2, ilog2ceil(pixelcount)) * sizeof(int));
	hipMalloc(&dev_paths_helper, pixelcount * sizeof(PathSegment));
	hipMalloc(&dev_paths_finished, pixelcount * sizeof(PathSegment));
    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    hipFree(dev_image);  // no-op if dev_image is null
  	hipFree(dev_paths);
  	hipFree(dev_geoms);
  	hipFree(dev_materials);
  	hipFree(dev_intersections);
    // TODO: clean up any extra device memory you created
	hipFree(dev_indices);
	hipFree(dev_bool);
	hipFree(dev_paths_helper);
    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment & segment = pathSegments[index];

		segment.ray.origin = cam.position;
    segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

		// TODO: implement antialiasing by jittering the ray
		thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
		thrust::uniform_real_distribution<float> u01(0.f, 1.0f);
		x += u01(rng);
		y += u01(rng);
		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
			);

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment * pathSegments
	, Geom * geoms
	, int geoms_size
	, ShadeableIntersection * intersections
	)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect; //World Space Intersection
		glm::vec3 tmp_normal; //World Space Normal

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom & geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			// TODO: add more intersection tests here... triangle? metaball? CSG?

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;
		}
	}
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial (
  int iter
  , int num_paths
	, ShadeableIntersection * shadeableIntersections
	, PathSegment * pathSegments
	, Material * materials
	)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_paths)
  {
    ShadeableIntersection intersection = shadeableIntersections[idx];
    if (intersection.t > 0.0f) { // if the intersection exists...
		// Set up the RNG
		// LOOK: this is how you use thrust's RNG! Please look at
		// makeSeededRandomEngine as well.
		
		thrust::uniform_real_distribution<float> u01(0, 1);

		Material material = materials[intersection.materialId];
		glm::vec3 materialColor = material.color;
		thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);

		// If the material indicates that the object was a light, "light" the ray
		if (material.emittance > 0.0f) {
			pathSegments[idx].color *= (materialColor * material.emittance);
		}
		// Otherwise, do some pseudo-lighting computation. This is actually more
		// like what you would expect from shading in a rasterizer like OpenGL.
		// TODO: replace this! you should be able to start with basically a one-liner
		else {
			float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(0.0f, 1.0f, 0.0f));
			pathSegments[idx].color *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
			pathSegments[idx].color *= u01(rng); // apply some noise because why not
		}
		// If there was no intersection, color the ray black.
		// Lots of renderers use 4 channel color, RGBA, where A = alpha, often
		// used for opacity, in which case they can indicate "no opacity".
		// This can be useful for post-processing and image compositing.
		} else {
			pathSegments[idx].color = glm::vec3(0.0f);
		}
	}
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 * image, PathSegment * iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

//Added by Yuxin
__global__ void kernBsdfShader(
	int iter
	, int num_paths, int depth
	, ShadeableIntersection * shadeableIntersections
	, PathSegment * pathSegments
	, Material * materials)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
	ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) { // if the intersection exists...
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
			PathSegment pathSegment = pathSegments[idx];
			Material m = materials[intersection.materialId];
			glm::vec3 intersect = pathSegment.ray.origin + pathSegment.ray.direction*intersection.t;
			scatterRay(pathSegments[idx], intersect, intersection.surfaceNormal, m, rng);
		}
		else {
			//no intersection
			pathSegments[idx].color = glm::vec3(0.0f);
			pathSegments[idx].remainingBounces = 0;
		}
	}
}

__global__ void kernMapRayToBoolean(int n, int *bools, const PathSegment *idata) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < n) {
		if (idata[index].remainingBounces <= 0) {
			bools[index] = 0;
		}
		else {
			bools[index] = 1;
		}
	}
}

__global__ void kernScatterRay(int n, PathSegment *odata,
	const PathSegment *idata, const int *bools, const int *indices) {
	// TODO
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < n) {
		if (bools[index] == 1) {
			odata[indices[index]] = idata[index];
		}
	}
}

__global__ void kernCalculateCompactResult(int numPath, const int *bools, int *indices) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < numPath) {
		//if (index == numPath - 1) {
			indices[index] += bools[index];
		//}
	}
}

__global__ void kernCopyFinishedPath(int numPath, PathSegment* oPath, const PathSegment* iPath) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < numPath) {
		if (iPath[index].remainingBounces <= 0) {
			oPath[iPath[index].pixelIndex] = iPath[index];
		}		
	}
}



/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4 *pbo, int frame, int iter) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;	

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * TODO: Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

    // TODO: perform one iteration of path tracing

	generateRayFromCamera <<<blocksPerGrid2d, blockSize2d >>>(cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");

	int depth = 0; //Question: do we use this to terminate the path tracing?
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths; //Questions: why not using pixelcount as num_paths???
	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks
	bool iterationComplete = false;
	hipMemset(dev_paths_finished, 0, pixelcount * sizeof(PathSegment));
	while (!iterationComplete) {

		// clean shading chunks
		hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));
		// clean stream compaction helper chunks
		hipMemset(dev_paths_helper, 0, pixelcount * sizeof(PathSegment));

		// tracing
		dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
		computeIntersections <<<numblocksPathSegmentTracing, blockSize1d>>> (
			depth
			, num_paths
			, dev_paths
			, dev_geoms
			, hst_scene->geoms.size()
			, dev_intersections
		);
		checkCUDAError("trace one bounce");
		hipDeviceSynchronize();
		depth++;

		// TODO:
		// --- Shading Stage ---
		// Shade path segments based on intersections and generate new rays by
		// evaluating the BSDF.
		// Start off with just a big kernel that handles all the different
		// materials you have in the scenefile.
		kernBsdfShader << <numblocksPathSegmentTracing, blockSize1d >> > (
			iter,
			num_paths,
			depth,
			dev_intersections,
			dev_paths,
			dev_materials
			);
		hipDeviceSynchronize();
		// TODO: compare between directly shading the path segments and shading
		// path segments that have been reshuffled to be contiguous in memory.

		/*shadeFakeMaterial<<<numblocksPathSegmentTracing, blockSize1d>>> (
			iter,
			num_paths,
			dev_intersections,
			dev_paths,
			dev_materials
		);*/
		
		//****************Copy current path Segment status to dev_paths_finished************//
		kernCopyFinishedPath << <numblocksPathSegmentTracing, blockSize1d >> > (num_paths, dev_paths_finished, dev_paths);
		hipDeviceSynchronize();
		//****************Copy current path Segment status to dev_paths_finished************//
		
		//****************Stream Compaction, remove the terminated ray***********************//
		hipMemset(dev_bool, num_paths, 0);
		hipMemset(dev_indices, num_paths, 0);
		bool isPowerOfTwo = (num_paths != 0) && ((num_paths & (num_paths - 1)) == 0);
		int npoweroftwo = num_paths;
		if (!isPowerOfTwo) {
			npoweroftwo = pow(2, ilog2ceil(num_paths));
		}
		kernMapRayToBoolean << <numblocksPathSegmentTracing, blockSize1d >> > (num_paths, dev_bool, dev_paths);
		hipDeviceSynchronize();
		//****************Stream Compaction Thrust******************//
		/*thrust::device_ptr<bool> boolFlag(dev_bool);
		thrust::device_ptr<PathSegment> path(dev_paths);
		thrust::remove_if(path, path + num_paths, boolFlag, thrust::logical_not<bool>());
		num_paths = thrust::count_if(boolFlag, boolFlag + num_paths, thrust::identity<bool>());*/
		//****************Stream Compaction Thrust******************//

		hipMemcpy(dev_indices, dev_bool, num_paths * sizeof(int), hipMemcpyDeviceToDevice);
		//Perform an exclusive sum scan on ibools to get the final indices array
		StreamCompaction::Efficient::gpuEfficientScan(num_paths, npoweroftwo, dev_indices);
		hipDeviceSynchronize();
		kernScatterRay << <numblocksPathSegmentTracing, blockSize1d >> > (num_paths, dev_paths_helper, dev_paths, dev_bool, dev_indices);
		hipDeviceSynchronize();

		kernCalculateCompactResult << <numblocksPathSegmentTracing, blockSize1d >> > (num_paths, dev_bool, dev_indices);
		hipDeviceSynchronize();
		int* indices = (int*)malloc(num_paths * sizeof(int));
		hipMemcpy(indices, dev_indices, num_paths * sizeof(int), hipMemcpyDeviceToHost);
		int compactResult = indices[num_paths-1];
		delete indices;
		num_paths = compactResult;
		hipMemset(dev_paths, 0, pixelcount * sizeof(PathSegment));
		hipMemcpy(dev_paths, dev_paths_helper, compactResult * sizeof(PathSegment), hipMemcpyDeviceToDevice);
		//****************Stream Compaction, remove the terminated ray***********************//		
		
		//If after stream compaction, no rays left in the pool, then set iterationComplete to true
		//iterationComplete = true; 
		// TODO: should be based off stream compaction results.
		if (num_paths < 1) {
		//Debug Purpose
		//if(depth==2){
			//Copy back the result from dev_paths_finished to dev_paths
			hipMemcpy(dev_paths, dev_paths_finished, pixelcount * sizeof(PathSegment), hipMemcpyDeviceToDevice);
			iterationComplete = true;
		}
	}

	// Assemble this iteration and apply it to the image
	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	num_paths = pixelcount;
	finalGather<<<numBlocksPixels, blockSize1d>>>(num_paths, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
            pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
