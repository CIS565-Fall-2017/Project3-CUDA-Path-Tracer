#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

//Include Stream Compaction files
#include "stream_compaction\efficient.h"


#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
    getchar();
#  endif
    exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
        int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

// =============================================================================
//					PATH TRACE INIT AND FREE CPU FUNCTIONS
// =============================================================================

static Scene * hst_scene = NULL;
static glm::vec3 * dev_image = NULL;
static Geom * dev_geoms = NULL;
static Material * dev_materials = NULL;
static PathSegment * dev_paths = NULL;
static ShadeableIntersection * dev_intersections = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...

//PathSeg and Isect indices sorted by Material
static int * dev_PathSegIndices = NULL;
static int * dev_IsectIndices = NULL;

//Caching first bounce
static ShadeableIntersection * dev_IsectCached = NULL;

void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

  	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

  	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
  	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

  	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
  	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

  	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
  	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    // TODO: initialize any extra device memeory you need

	//PathSeg and Isect indices sorted by Material
	hipMalloc(&dev_PathSegIndices, pixelcount * sizeof(int));

	hipMalloc(&dev_IsectIndices, pixelcount * sizeof(int));

	//Caching first bounce
	hipMalloc(&dev_IsectCached, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_IsectCached, 0, pixelcount * sizeof(ShadeableIntersection));

    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    hipFree(dev_image);  // no-op if dev_image is null
  	hipFree(dev_paths);
  	hipFree(dev_geoms);
  	hipFree(dev_materials);
  	hipFree(dev_intersections);
    // TODO: clean up any extra device memory you created

	hipFree(dev_PathSegIndices);
	hipFree(dev_IsectIndices);
	hipFree(dev_IsectCached);

    checkCUDAError("pathtraceFree");
}

// =============================================================================
//				GENERATE 3D POINT BASED ON DISK CONOCENTRIC
// =============================================================================

__host__ __device__
glm::vec3 squareToDiskConocentric(const glm::vec2 &sample)
{
	glm::vec3 output(0.0f);
	glm::vec2 uOffset = 2.0f * sample - glm::vec2(1.0f);

	if (uOffset.x == 0.0f + RAY_EPSILON && uOffset.y == 0.0f + RAY_EPSILON)
	{
		return glm::vec3(0.0f);
	}

	float theta = 0.0f;
	float r = 0.0f;

	if (std::fabs(uOffset.x) > std::fabs(uOffset.y))
	{
		r = uOffset.x;
		theta = (PI / 4.0f) *  (uOffset.y / uOffset.x);
	}

	else
	{
		r = uOffset.y;
		theta = (PI / 2.0f) - (PI / 4.0f) * (uOffset.x / uOffset.y);
	}

	output = r * glm::vec3(cos(theta), sin(theta), 0.0f);
	return output;
}


// =============================================================================
//					GENERATE RAY FROM CAMERA KERNEL FUNCTION
// =============================================================================

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) 
	{
		int index = x + (y * cam.resolution.x);
		PathSegment & segment = pathSegments[index];

		segment.ray.origin = cam.position;
		segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

		// TODO: implement antialiasing by jittering the ray
		// Note: If antialiasing -- can NOT cache first bounce!

		thrust::default_random_engine rng = makeSeededRandomEngine(iter, x, y);
		thrust::uniform_real_distribution<float> u01(-1, 1);
		float offset_x = u01(rng);
		float offset_y = u01(rng);

		//Non anti-aliased
		segment.ray.direction = glm::normalize(cam.view
												- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
												- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
												);
		
		//Anti-Aliasing
		//segment.ray.direction = glm::normalize(cam.view
		//										- cam.right * cam.pixelLength.x * ((float)x + offset_x - (float)cam.resolution.x * 0.5f)
		//										- cam.up * cam.pixelLength.y * ((float)y + offset_y - (float)cam.resolution.y * 0.5f)
		//										);

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;

		//Depth of field
		//Generate sample point on disk lens, shoot ray through that
		if (cam.lensRadius > 0.0f)
		{
			thrust::uniform_real_distribution<float> u02(0, 1);
			glm::vec2 sample = glm::vec2(u02(rng), u02(rng));

			glm::vec3 pLens = cam.lensRadius * squareToDiskConocentric(sample);
			glm::vec3 pFocus = cam.focalDistance * segment.ray.direction + segment.ray.origin;
			glm::vec3 aperaturePt = segment.ray.origin + (cam.up * pLens[1]) + (cam.right * pLens[0]);

			segment.ray.origin = aperaturePt;
			segment.ray.direction = glm::normalize(pFocus - aperaturePt);
		}//end DOF check

	}//end if 
}

// =============================================================================
//						COMPUTE INTERSECTION KERNEL FUNCTION
// =============================================================================

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment * pathSegments
	, Geom * geoms
	, int geoms_size
	, ShadeableIntersection * intersections
	)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom & geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			// TODO: add more intersection tests here... triangle? metaball? CSG?

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}//end for all geoms

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;			//QUESTION: should this be normalized?
			//intersections[path_index].surfaceNormal = glm::normalize(normal);
			intersections[path_index].intersectionPt = intersect_point;
		}
	}
}


// =============================================================================
//							SHADER KERNEL FUNCTIONS
// =============================================================================


// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial (
  int iter
  , int num_paths
	, ShadeableIntersection * shadeableIntersections
	, PathSegment * pathSegments
	, Material * materials
	, int depth
	)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		if (pathSegments[idx].remainingBounces <= 0) 
		{
			return;
		}

		ShadeableIntersection intersection = shadeableIntersections[idx];
		
		// if the intersection exists...
		if (intersection.t > 0.0f) 
		{ 
			// Set up the RNG
			// LOOK: this is how you use thrust's RNG! Please look at makeSeededRandomEngine as well.

			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
			
			//Use depth for non-compact version to work properly
			//https://groups.google.com/forum/#!topic/cis-565-fall-2017/thgdf2jzDyo
			//thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, depth);
			//thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, pathSegments[idx].remainingBounces);


			thrust::uniform_real_distribution<float> u01(0, 1);

			Material material = materials[intersection.materialId];
			glm::vec3 materialColor = material.color;

			// If the material indicates that the object was a light, "light" the ray
			if (material.emittance > 0.0f) 
			{
				pathSegments[idx].color *= (materialColor * material.emittance);
				pathSegments[idx].remainingBounces = 0;
			}
			// Otherwise, do some pseudo-lighting computation. This is actually more
			// like what you would expect from shading in a rasterizer like OpenGL.
			// TODO: replace this! you should be able to start with basically a one-liner
			else 
			{
				//float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(0.0f, 1.0f, 0.0f));
				//pathSegments[idx].color *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
				//pathSegments[idx].color *= u01(rng); // apply some noise because why not

				scatterRay(pathSegments[idx], intersection.intersectionPt, intersection.surfaceNormal, material, rng);
				pathSegments[idx].remainingBounces--;

			}
			// If there was no intersection, color the ray black.
			// Lots of renderers use 4 channel color, RGBA, where A = alpha, often
			// used for opacity, in which case they can indicate "no opacity".
			// This can be useful for post-processing and image compositing.
		}//end if

		else 
		{
			pathSegments[idx].color = glm::vec3(0.0f);
			pathSegments[idx].remainingBounces = 0;
		}//end else
	}
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 * image, PathSegment * iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}



// =============================================================================
//							PATH TERMINATION SCAN KERNEL
// =============================================================================

// Check if remaining bounces == 0
// Check if path intersection t value == -1 (didn't hit anything)

//UNUSED
__global__ void kernMapRemainingBouncesToBoolean(int n, int *bools, PathSegment *pathSegments)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < n)
	{
		//If path's remanining bounces is > 0, mark as 1, else 0
		PathSegment currPath = pathSegments[index];
		if (currPath.remainingBounces > 0)		bools[index] = 1;
		else									bools[index] = 0;
	}
}//end kernMapRemainingBounces

//UNUSED
__global__ void kernMapNoIsectPathToBoolean(int n, int *bools, ShadeableIntersection *intersections)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < n)
	{
		ShadeableIntersection currIsect = intersections[index];
		//if()
	}
}

// Predicate for thrust::partition
struct hasRemainingBounces
{
	__host__ __device__
	bool operator()(const PathSegment &pathSegment)
	{
		return pathSegment.remainingBounces > 0;
	}
};


//Fill dev_PathSegIndices and dev_IsectIndices with their corresponding material ID
// These arrays should essentially be the same since pathSeg's and Isects correspond to each other
__global__ void kernSortByMaterial(int n, int *pathSegIndices, int *isectIndices, ShadeableIntersection *isects)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < n)
	{
		int currMatID = isects[index].materialId;
		pathSegIndices[index] = currMatID;
		isectIndices[index] = currMatID;
	}
}

// =============================================================================
//							PATH TRACING CPU FUNCTION
// =============================================================================

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4 *pbo, int frame, int iter) 
{
	const int traceDepth = hst_scene->state.traceDepth;
	const Camera &cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
								(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
								(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

	///////////////////////////////////////////////////////////////////////////

	// Recap:
	// * Initialize array of path rays (using rays that come out of the camera)
	//   * You can pass the Camera object to that kernel.
	//   * Each path ray must carry at minimum a (ray, color) pair,
	//   * where color starts as the multiplicative identity, white = (1, 1, 1).
	//   * This has already been done for you.
	// * For each depth:
	//   * Compute an intersection in the scene for each path ray.
	//     A very naive version of this has been implemented for you, but feel
	//     free to add more primitives and/or a better algorithm.
	//     Currently, intersection distance is recorded as a parametric distance,
	//     t, or a "distance along the ray." t = -1.0 indicates no intersection.
	//     * Color is attenuated (multiplied) by reflections off of any object
	//   * TODO: Stream compact away all of the terminated paths.
	//     You may use either your implementation or `thrust::remove_if` or its
	//     cousins.
	//     * Note that you can't really use a 2D kernel launch any more - switch
	//       to 1D.
	//   * TODO: Shade the rays that intersected something or didn't bottom out.
	//     That is, color the ray by performing a color computation according
	//     to the shader, then generate a new ray to continue the ray path.
	//     We recommend just updating the ray's PathSegment in place.
	//     Note that this step may come before or after stream compaction,
	//     since some shaders you write may also cause a path to terminate.
	// * Finally, add this iteration's results to the image. This has been done
	//   for you.

	// TODO: perform one iteration of path tracing

	generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;

	int num_remainingPaths = num_paths;


	//NOTE: USE TIMER TO PRINT THE TIME FOR EACH ITERATION OF WHILE LOOP


	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

	bool iterationComplete = false;
	while (!iterationComplete) 
	{
		// clean shading chunks
		hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

		//DO I NEED THIS? OR DO I NOT WANT TO DO THIS BECAUSE I WANT TO KEEP USING IT?
		//hipMemset(dev_IsectCached, 0, pixelcount * sizeof(ShadeableIntersection));

		// tracing
		//dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
		dim3 numblocksPathSegmentTracing = (num_remainingPaths + blockSize1d - 1) / blockSize1d;

		//Caching first bounce
		//Don't start at iter = 0, that's ray from camera to screen
		
		// Compute intersections

		if (CACHE_FIRST_BOUNCE && depth == 0 && iter == 1)
		{
			computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>>(
																				depth
																				, num_remainingPaths //num_paths
																				, dev_paths
																				, dev_geoms
																				, hst_scene->geoms.size()
																				, dev_IsectCached
																				);
		}//end if
		else
		{
			computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>>(
																				depth
																				, num_remainingPaths //num_paths
																				, dev_paths
																				, dev_geoms
																				, hst_scene->geoms.size()
																				, dev_intersections
																				);
		}//end else

		//TESTING 
		//computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>>(
		//	depth
		//	, num_remainingPaths //num_paths
		//	, dev_paths
		//	, dev_geoms
		//	, hst_scene->geoms.size()
		//	, dev_intersections
		//	);


		checkCUDAError("trace one bounce");
		hipDeviceSynchronize();
		//depth++;


		// TODO:
		// --- Shading Stage ---
		// Shade path segments based on intersections and generate new rays by
		// evaluating the BSDF.
		// Start off with just a big kernel that handles all the different
		// materials you have in the scenefile.
		// TODO: compare between directly shading the path segments and shading
		// path segments that have been reshuffled to be contiguous in memory.

		if (CACHE_FIRST_BOUNCE && depth == 0 && iter == 1)
		{
			if (SORT_BY_MATERIAL)
			{
				//Store material ID's in dev_PathSegIndices and dev_IsectIndices respectively
				kernSortByMaterial<<<numblocksPathSegmentTracing, blockSize1d>>>(num_remainingPaths, dev_PathSegIndices, dev_IsectIndices, dev_IsectCached);

				//Sort the PathSegments and Isects arrays in place according to materialID's placed in their corresponding dev_indices arrays 
				thrust::sort_by_key(thrust::device, dev_PathSegIndices, dev_PathSegIndices + num_remainingPaths, dev_paths);
				thrust::sort_by_key(thrust::device, dev_IsectIndices, dev_IsectIndices + num_remainingPaths, dev_IsectCached);
			}

			shadeFakeMaterial<<<numblocksPathSegmentTracing, blockSize1d>>>(
																			iter,
																			num_remainingPaths, //num_paths,
																			dev_IsectCached,
																			dev_paths,
																			dev_materials,
																			depth
																			);
		}//end if first bounce

		//Operating on everything else after first bounce
		else
		{
			if (SORT_BY_MATERIAL)
			{
				//Store material ID's in dev_PathSegIndices and dev_IsectIndices respectively
				kernSortByMaterial<<<numblocksPathSegmentTracing, blockSize1d>>>(num_remainingPaths, dev_PathSegIndices, dev_IsectIndices, dev_intersections);

				//Sort the PathSegments and Isects arrays in place according to materialID's placed in their corresponding dev_indices arrays 
				thrust::sort_by_key(thrust::device, dev_PathSegIndices, dev_PathSegIndices + num_remainingPaths, dev_paths);
				thrust::sort_by_key(thrust::device, dev_IsectIndices, dev_IsectIndices + num_remainingPaths, dev_intersections);
			}


			shadeFakeMaterial<<<numblocksPathSegmentTracing, blockSize1d>>>(
																			iter,
																			num_remainingPaths, //num_paths,
																			dev_intersections,
																			dev_paths,
																			dev_materials,
																			depth
																			);
		}//end else 

		//TESTING
		//shadeFakeMaterial<<<numblocksPathSegmentTracing, blockSize1d>>>(
		//																iter,
		//																num_remainingPaths, //num_paths,
		//																dev_intersections,
		//																dev_paths,
		//																dev_materials,
		//																depth
		//																);


		// TODO: Stream compact away all of the terminated paths.
		// You may use either your implementation or `thrust::remove_if` or its cousins.
		// NOTE: DO NOT USE REMOVE_IF
		PathSegment* lastRemainingPath = thrust::partition(thrust::device, dev_paths, dev_paths + num_remainingPaths, hasRemainingBounces());
		num_remainingPaths = lastRemainingPath - dev_paths;

		// Increment depth
		depth++;


		// TODO: should be based off stream compaction results.
		// To test anti-aliasing, change depth >= 1, and move the camera around. You'll see jagged edges become smoother
		iterationComplete = ((depth >= traceDepth || num_remainingPaths <= 0) ? true : false);

	}//end while

	  // Assemble this iteration and apply it to the image
	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather<<<numBlocksPixels, blockSize1d>>>(num_paths, dev_image, dev_paths);

	///////////////////////////////////////////////////////////////////////////

	// Send results to OpenGL buffer for rendering
	sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

	// Retrieve image from GPU
	hipMemcpy(hst_scene->state.image.data(), dev_image,
				pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	checkCUDAError("pathtrace");
}//end pathTrace
