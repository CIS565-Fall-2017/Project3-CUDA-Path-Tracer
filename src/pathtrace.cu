#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"
#include "build\src\FresnelDielectric.h"


#define lensRadius 1.f
#define focalDistance 1.f
#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
    getchar();
#  endif
    exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
        int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene * hst_scene = NULL;
static glm::vec3 * dev_image = NULL;
static Geom * dev_geoms = NULL;
static Material * dev_materials = NULL;
static PathSegment * dev_paths = NULL;
static ShadeableIntersection * dev_intersections = NULL;
static int* compactSteamsIn= NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...

void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

  	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

  	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
  	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

  	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
  	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

  	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
  	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    // TODO: initialize any extra device memeory you need

	hipMalloc(&compactSteamsIn, pixelcount * sizeof(int));


    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    hipFree(dev_image);  // no-op if dev_image is null
  	hipFree(dev_paths);
  	hipFree(dev_geoms);
  	hipFree(dev_materials);
  	hipFree(dev_intersections);
    // TODO: clean up any extra device memory you created

	hipFree(compactSteamsIn);

    checkCUDAError("pathtraceFree");
}

__device__ void RealisticCamera(PathSegment& pathsegment, thrust::default_random_engine &rng)
{
	glm::vec3 pinholeRayOri = pathsegment.ray.origin;
	glm::vec3 pinholeRayDir = pathsegment.ray.direction;
	glm::vec3 rayOrigin = glm::vec3(0.f);
	glm::vec3 rayDirection = glm::vec3(0.f);

	thrust::uniform_real_distribution<float> u01(0, 1);
	thrust::uniform_real_distribution<float> u02(0, 1);
	float samplex = u01(rng);
	float sampley = u02(rng);

	if (lensRadius > 0)
	{
		glm::vec3 pLens = lensRadius * squareToDiskConcentric(glm::vec2(samplex,sampley));

		float ft = (focalDistance - pinholeRayOri.z) / pinholeRayDir.z;
		glm::vec3 pFocus = pinholeRayOri + pinholeRayDir * ft;


		rayOrigin = pinholeRayOri + pLens;
		rayDirection = glm::normalize(pFocus - rayOrigin);
	}
	pathsegment.ray.origin = rayOrigin;
	pathsegment.ray.direction = rayDirection;

}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment & segment = pathSegments[index];

		segment.ray.origin = cam.position;
    segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

		// TODO: implement antialiasing by jittering the ray
		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
			);

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
		

		//LOOK:special code for realistic camera
		//thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
		//RealisticCamera(pathSegments[index], rng);
	}

	
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment * pathSegments
	, Geom * geoms
	, int geoms_size
	, ShadeableIntersection * intersections
	)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom & geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			// TODO: add more intersection tests here... triangle? metaball? CSG?

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;
		}
	}
}

__global__ void shadeMaterial(
	int iter
	, int num_paths
	, ShadeableIntersection * shadeableIntersections
	, PathSegment * pathSegments
	, Material * materials
	,int depth
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (glm::length(pathSegments[idx].color - glm::vec3(0.f)) <= FLT_EPSILON)
	{
		pathSegments[idx].remainingBounces = 0;
		return;
	}

	if (idx < num_paths)
	{
		int bounceDepth = depth - pathSegments[idx].remainingBounces;
		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) { // if the intersection exists...
									 // Set up the RNG
									 // LOOK: this is how you use thrust's RNG! Please look at
									 // makeSeededRandomEngine as well.
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, bounceDepth);
			thrust::uniform_real_distribution<float> u01(0, 1);

			Material material = materials[intersection.materialId];
			glm::vec3 materialColor = material.color;

			// If the material indicates that the object was a light, "light" the ray
			if (material.emittance > 0.0f) {
				pathSegments[idx].color *= (materialColor * material.emittance);
				pathSegments[idx].remainingBounces = 0;
			}
			else {
				glm::vec3 lastIntersectionPoint = pathSegments[idx].ray.origin;
				glm::vec3 lastIntersectionDir = pathSegments[idx].ray.direction;
				glm::vec3 intersectionPoint = lastIntersectionPoint + lastIntersectionDir*intersection.t;
				scatterRay(pathSegments[idx], intersectionPoint, intersection.surfaceNormal, material, rng);
				pathSegments[idx].remainingBounces--;
			}
		}
		else {
			pathSegments[idx].color = glm::vec3(0.0f);
			pathSegments[idx].remainingBounces = 0;
		}
	}
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial (
  int iter
  , int num_paths
	, ShadeableIntersection * shadeableIntersections
	, PathSegment * pathSegments
	, Material * materials
	)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (glm::length(pathSegments[idx].color - glm::vec3(0.f)) <= FLT_EPSILON)
  {
	  pathSegments[idx].remainingBounces = 0;
	  return;
  }

  if (idx < num_paths)
  {
    ShadeableIntersection intersection = shadeableIntersections[idx];
    if (intersection.t > 0.0f) { // if the intersection exists...
      // Set up the RNG
      // LOOK: this is how you use thrust's RNG! Please look at
      // makeSeededRandomEngine as well.
      thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
      thrust::uniform_real_distribution<float> u01(0, 1);

      Material material = materials[intersection.materialId];
      glm::vec3 materialColor = material.color;

      // If the material indicates that the object was a light, "light" the ray
      if (material.emittance > 0.0f) {
        pathSegments[idx].color *= (materialColor * material.emittance);
		pathSegments[idx].remainingBounces=0;
      }
      // Otherwise, do some pseudo-lighting computation. This is actually more
      // like what you would expect from shading in a rasterizer like OpenGL.
      // TODO: replace this! you should be able to start with basically a one-liner
      else {
        //float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(0.0f, 1.0f, 0.0f));
        //pathSegments[idx].color *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
        //pathSegments[idx].color *= u01(rng); // apply some noise because why not

		  glm::vec3 lastIntersectionPoint = pathSegments[idx].ray.origin;
		  glm::vec3 lastIntersectionDir = pathSegments[idx].ray.direction;
		  glm::vec3 intersectionPoint = lastIntersectionPoint + lastIntersectionDir*intersection.t;
		  scatterRay(pathSegments[idx], intersectionPoint, intersection.surfaceNormal, material, rng);
		  pathSegments[idx].remainingBounces--;
      }
    // If there was no intersection, color the ray black.
    // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
    // used for opacity, in which case they can indicate "no opacity".
    // This can be useful for post-processing and image compositing.
    } else {
      pathSegments[idx].color = glm::vec3(0.0f);
	  pathSegments[idx].remainingBounces = 0;
    }
  } 
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 * image, PathSegment * iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		//glm::vec3 colorToShow = Clamp(iterationPath.color, 0.f, 1.f);
		glm::vec3 colorToShow = glm::clamp(iterationPath.color, 0.f, 3.f);
		image[iterationPath.pixelIndex] += colorToShow;
	}
}

__global__ void BouncesLeft(PathSegment * pathSegments, int* steamCompactIn, int pixelCount)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < pixelCount)
	{
		PathSegment thisPath = pathSegments[index];
		steamCompactIn[index] = thisPath.remainingBounces;
	}
}

__device__ int DirectShadowIntersection(
	PathSegment & pathSegment
	, Geom * geoms
	, int geoms_size
	, ShadeableIntersection & intersection)
{
		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom & geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			// TODO: add more intersection tests here... triangle? metaball? CSG?

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		
		//material 0 means hit the light source
		if (hit_geom_index == -1)
		{
			intersection.t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersection.t = t_min;
			intersection.materialId = geoms[hit_geom_index].materialid;
			intersection.surfaceNormal = normal;
		}
				
}

//TODO My own Path tracer DirectLighting pa
__global__ void DirectLightingIntegrator(
	int iter
	, int num_paths
	, ShadeableIntersection * shadeableIntersections
	, PathSegment * pathSegments
	, Material * materials
	, Geom * geoms
    , int lightCount)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < num_paths)
	{
		PathSegment pathSegment = pathSegments[idx];
		ShadeableIntersection intersection = shadeableIntersections[idx];
		Material materialIsc = materials[intersection.materialId];
		glm::vec3 leColor = materialIsc.color;

		thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
		thrust::uniform_real_distribution<float> u01(0, 1);

		//meets the light source
		if (materialIsc.emittance > 0.f)
		{
			pathSegments[idx].color = leColor;
		}
		else
		{
			int chosenLight = (int)(u01(rng)*lightCount);
			glm::vec3 liColor = materials[0].color;
		}
	}
}


/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4 *pbo, int frame, int iter) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;
	const int blocksPerGrid1d = (pixelcount + blockSize1d - 1) / blockSize1d;

	int* steamCompactionOut;
	steamCompactionOut = (int*)malloc(sizeof(int)*pixelcount);
	int *newEnd = NULL;

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * TODO: Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust`::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

    // TODO: perform one iteration of path tracing

	generateRayFromCamera <<<blocksPerGrid2d, blockSize2d >>>(cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;
	int compactSize = 0;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

  bool iterationComplete = false;
  dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	while (!iterationComplete) {

	// clean shading chunks
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	// tracing
	dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
	computeIntersections <<<numblocksPathSegmentTracing, blockSize1d>>> (
		depth
		, num_paths
		, dev_paths
		, dev_geoms
		, hst_scene->geoms.size()
		, dev_intersections
		);
	checkCUDAError("trace one bounce");
	hipDeviceSynchronize();
	depth++;


	// TODO:
	// --- Shading Stage ---
	// Shade path segments based on intersections and generate new rays by
  // evaluating the BSDF.
  // Start off with just a big kernel that handles all the different
  // materials you have in the scenefile.
  // TODO: compare between directly shading the path segments and shading
  // path segments that have been reshuffled to be contiguous in memory.

  //shadeFakeMaterial<<<numblocksPathSegmentTracing, blockSize1d>>> (
  //  iter,
  //  num_paths,
  //  dev_intersections,
  //  dev_paths,
  //  dev_materials
  //);
	shadeMaterial << <numblocksPathSegmentTracing, blockSize1d >> > (
		iter,
		num_paths,
		dev_intersections,
		dev_paths,
		dev_materials,
		traceDepth);

	
	//finalGather << <numBlocksPixels, blockSize1d >> >(num_paths, dev_image, dev_paths);
  
  //TODO I added
  BouncesLeft << <blocksPerGrid1d, blockSize1d >> > (dev_paths, compactSteamsIn, pixelcount);
  hipMemcpy(steamCompactionOut, compactSteamsIn, sizeof(int)*pixelcount, hipMemcpyDeviceToHost);

  newEnd = thrust::remove(steamCompactionOut, steamCompactionOut + pixelcount, 0);
  

  if (newEnd == steamCompactionOut)
  {
	  iterationComplete = true; // TODO: should be based off stream compaction results.
  }
  
	}

	free(steamCompactionOut);
  // Assemble this iteration and apply it to the image
  /*dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;*/
	finalGather<<<numBlocksPixels, blockSize1d>>>(num_paths, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
            pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
