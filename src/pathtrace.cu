#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/sort.h>
#include <thrust/device_vector.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 0
#define SORT_MATERIALS 0
#define CACHE_PATHS 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
    getchar();
#  endif
    exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
        int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene * hst_scene = NULL;
static glm::vec3 * dev_image = NULL;
static Geom * dev_geoms = NULL;
static Material * dev_materials = NULL;
static PathSegment* dev_cached_paths = NULL;
static PathSegment * dev_paths = NULL;
static ShadeableIntersection * dev_cached_intersections = NULL;
static ShadeableIntersection * dev_intersections = NULL;

thrust::device_ptr<PathSegment> dev_thrust_paths;
thrust::device_ptr<ShadeableIntersection> dev_thrust_intersections;

// TODO: static variables for device memory, any extra info you need, etc
// ...

void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

  	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));
	hipMalloc(&dev_cached_paths, pixelcount * sizeof(PathSegment));

  	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
  	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

  	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
  	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);
	
	hipMalloc(&dev_cached_intersections, pixelcount * sizeof(ShadeableIntersection));
  	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
  	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	dev_thrust_paths = thrust::device_ptr<PathSegment>(dev_paths);
	dev_thrust_intersections = thrust::device_ptr<ShadeableIntersection>(dev_intersections);

    // TODO: initialize any extra device memeory you need

    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    hipFree(dev_image);  // no-op if dev_image is null
  	hipFree(dev_paths);
  	hipFree(dev_geoms);
  	hipFree(dev_materials);
  	hipFree(dev_intersections);
    // TODO: clean up any extra device memory you created

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment & segment = pathSegments[index];

		segment.ray.origin = cam.position;
		segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

		thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
		thrust::uniform_real_distribution<float> u01(0, 1);

		// TODO: implement antialiasing by jittering the ray
		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x + u01(rng) - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y + u01(rng) - (float)cam.resolution.y * 0.5f)
			);

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment * pathSegments
	, Geom * geoms
	, int geoms_size
	, ShadeableIntersection * intersections
	)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom & geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == TRIANGLE) {
				t = triangleIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			// TODO: add more intersection tests here... triangle? metaball? CSG?

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}
		
		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
			pathSegments[path_index].remainingBounces = 0;
		}
		else
		{
			//The ray hits something
			//pathSegment.remainingBounces--;
			intersections[path_index].t = t_min;
			intersections[path_index].point = intersect_point;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;
			pathSegments[path_index].insideT = outside ? 0.0f : t_min;
		}
	}
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial(
	int iter
	, int num_paths
	, ShadeableIntersection * shadeableIntersections
	, PathSegment * pathSegments
	, Material * materials
	, glm::vec3 *image
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) { // if the intersection exists...
									 // Set up the RNG
									 // LOOK: this is how you use thrust's RNG! Please look at
									 // makeSeededRandomEngine as well.
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
			thrust::uniform_real_distribution<float> u01(0, 1);

			Material material = materials[intersection.materialId];
			glm::vec3 materialColor = material.color;

			// If the material indicates that the object was a light, "light" the ray
			if (material.emittance > 0.0f) {
				image[pathSegments[idx].pixelIndex] += pathSegments[idx].color * (materialColor * material.emittance);
				pathSegments[idx].remainingBounces = 0;
			}
			// Otherwise, color and bounce
			else {
				scatterRay(pathSegments[idx], intersection.point, intersection.surfaceNormal, material, rng);
#if DISPLAY_NORMALS
				image[pathSegments[idx].pixelIndex] += pathSegments[idx].color;
#endif
				pathSegments[idx].remainingBounces--;
			}
			// If there was no intersection, color the ray black.
			// Lots of renderers use 4 channel color, RGBA, where A = alpha, often
			// used for opacity, in which case they can indicate "no opacity".
			// This can be useful for post-processing and image compositing.
		}
		else {
			//image[pathSegments[idx].pixelIndex] += glm::vec3(0.0f);
			//pathSegments[idx].color = glm::vec3(0.0f);
			pathSegments[idx].remainingBounces = 0;
		}
	}
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 * image, PathSegment * iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

//from https://thrust.github.io/doc/group__stream__compaction.html#ga307d7f64566909172a3f9e16b7e2ad53
struct path_complete {
	__host__ __device__ bool operator()(PathSegment p) {
		return p.remainingBounces <= 0;
	}
};

//from http://www.sgi.com/tech/stl/StrictWeakOrdering.html and the above link
struct matComparator
{
	__host__ __device__ bool operator()(ShadeableIntersection &isThis, ShadeableIntersection &lessThanThis)
	{
		return isThis.materialId < lessThanThis.materialId;
	}
};

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4 *pbo, int frame, int iter) {
	const int traceDepth = hst_scene->state.traceDepth;
	const Camera &cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;
	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;

	//perform first bounce OR load cached first bounce
#if CACHE_PATHS
	//store the initial paths and intersections, which will remain constant until camera moves (at which point iter == 0)
	//currently jitters rays every 25 passes for antialiasing
	if (iter % 25 == 1) {
		generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth, dev_cached_paths);
		computeIntersections << <numBlocksPixels, blockSize1d >> > (0, pixelcount, dev_cached_paths, dev_geoms, 
			hst_scene->geoms.size(), dev_cached_intersections);
		hipDeviceSynchronize();
	}
	hipMemcpy(dev_paths, dev_cached_paths, pixelcount * sizeof(PathSegment), hipMemcpyDefault);
	hipMemcpy(dev_intersections, dev_cached_intersections, pixelcount * sizeof(ShadeableIntersection), hipMemcpyDefault);
#else
	generateRayFromCamera <<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths);
	computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
		depth
		, num_paths
		, dev_paths
		, dev_geoms
		, hst_scene->geoms.size()
		, dev_cached_intersections
		);
#endif
	checkCUDAError("First Bounce");

	int depth = 1;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;
	bool iterationComplete = false;
	dim3 numblocksPathSegmentTracing = numBlocksPixels;

	while (!iterationComplete) {

#if SORT_MATERIALS
		thrust::sort_by_key(dev_thrust_intersections, dev_thrust_intersections + num_paths, dev_thrust_paths, matComparator());
		checkCUDAError("Failed to Sort Material IDs");
#endif

		//SHADE INTERSECTIONS, SCATTER RAYS
		shadeFakeMaterial << <numblocksPathSegmentTracing, blockSize1d >> > (iter, num_paths, dev_intersections, dev_paths,
			dev_materials, dev_image);

		//STREAM COMPACT TO REMOVE USELESS PATHS
		PathSegment* new_dev_paths_end = thrust::remove_if(thrust::device, dev_paths, dev_paths + num_paths, path_complete());//-- 2: cull those paths that don't need any more shading
		num_paths = new_dev_paths_end - dev_paths;

		//END THIS ITERATION IF WE'VE FINISHED NEARLY EVERY PATH, OR ELSE COMPUTE NEW INTERSECTIONS
		if (num_paths == 0) iterationComplete = true;
		else {
			hipMemset(dev_intersections, 0, num_paths * sizeof(ShadeableIntersection));
			// tracing
			numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
			computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (depth, num_paths, dev_paths, dev_geoms, 
				hst_scene->geoms.size(), dev_intersections);
			checkCUDAError("trace one bounce");
			hipDeviceSynchronize();
			depth++;
		}
	}

	//FINAL GATHER
	finalGather << <numBlocksPixels, blockSize1d >> >(num_paths, dev_image, dev_paths); //----------------------------------------- add final contributions to the frame

	///////////////////////////////////////////////////////////////////////////

	// Send results to OpenGL buffer for rendering
	sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_image);

	// Retrieve image from GPU
	hipMemcpy(hst_scene->state.image.data(), dev_image,
		pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	checkCUDAError("pathtrace");
}
