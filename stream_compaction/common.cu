#include "hip/hip_runtime.h"
#include "common.h"

void checkCUDAErrorFn(const char *msg, const char *file, int line) {
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
}


namespace StreamCompaction {
    namespace Common {

        /**
         * Maps an array to an array of 0s and 1s for stream compaction. Elements
         * which map to 0 will be removed, and elements which map to 1 will be kept.
         */
        __global__ void kernMapToBoolean(int n, int *bools, const int *idata) {
            // TODO
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;
			if (index >= n || idata[index] == 0) return;

			bools[index] = 1;
        }

        /**
         * Performs scatter on an array. That is, for each element in idata,
         * if bools[idx] == 1, it copies idata[idx] to odata[indices[idx]].
         */
        __global__ void kernScatter(int n, int *odata, const int *idata, const int *indices) {
            // TODO
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;
 			if (index >= n || idata[index] == 0) return;

 			odata[indices[index]] = idata[index];
        }
    }
}