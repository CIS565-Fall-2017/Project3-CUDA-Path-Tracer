#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
			// Do not time memory allocation
			int* dev_odata;
			hipMalloc((void**)&dev_odata, n * sizeof(int));
			int* dev_idata;
			hipMalloc((void**)&dev_idata, n * sizeof(int));
			hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
			thrust::device_ptr<int> dv_out(dev_odata);
			thrust::device_ptr<int> dv_in(dev_idata);
            
			// Time everything else
			timer().startGpuTimer();
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
			thrust::exclusive_scan(dv_in, dv_in + n, dv_out);
            timer().endGpuTimer();

			// Get the return value off of the device and free memory.
			hipMemcpy(odata, dev_odata, sizeof(int) * n, hipMemcpyDeviceToHost);
			hipFree(dev_odata);
			hipFree(dev_idata);
        }
    }
}
