#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
			int* dv_in, *dv_out;
			hipMalloc((void**)&dv_in, sizeof(int) * n);
			hipMalloc((void**)&dv_out, sizeof(int) * n);

			hipMemcpy(dv_in, idata, sizeof(int) * n, hipMemcpyHostToDevice);

			thrust::device_ptr<int> dv_in_thrust(dv_in);
			thrust::device_ptr<int> dv_out_thrust(dv_out);
			thrust::exclusive_scan(dv_in_thrust, dv_in_thrust + n, dv_out_thrust);

            timer().startGpuTimer();
			thrust::exclusive_scan(dv_in_thrust, dv_in_thrust + n, dv_out_thrust);
            timer().endGpuTimer();

			hipMemcpy(odata, dv_out, sizeof(int) * n, hipMemcpyDeviceToHost);

			hipFree(dv_in);
			hipFree(dv_out);
        }
    }
}
