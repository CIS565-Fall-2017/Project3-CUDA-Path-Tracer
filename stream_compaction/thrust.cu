#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
			//https://stackoverflow.com/questions/9495599/thrust-how-to-create-device-vector-from-host-array
			thrust::device_vector<int> dv_in(idata, idata + n);
			thrust::device_vector<int> dv_out(odata, odata + n);
            timer().startGpuTimer();
			thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
            timer().endGpuTimer();
			thrust::copy(dv_out.begin(), dv_out.end(), odata);
        }
    }
}
