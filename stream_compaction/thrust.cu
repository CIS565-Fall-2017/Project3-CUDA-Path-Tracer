#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
			int *dev_idata;
			int *dev_odata;
			
			hipMalloc((void**)&dev_idata, n * sizeof(int));
			checkCUDAError("hipMalloc dev_idata failed!");

			hipMalloc((void**)&dev_odata, n * sizeof(int));
			checkCUDAError("hipMalloc dev_odata failed!");

			hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy failed!");

			thrust::device_ptr<int> dev_thrust_idata(dev_idata);
			thrust::device_ptr<int> dev_thrust_odata(dev_odata);

            timer().startGpuTimer();
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());

			thrust::exclusive_scan(dev_thrust_idata, dev_thrust_idata + n, dev_thrust_odata);
            timer().endGpuTimer();

			hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);

			hipFree(dev_idata);
			hipFree(dev_odata);
        }
    }
}
