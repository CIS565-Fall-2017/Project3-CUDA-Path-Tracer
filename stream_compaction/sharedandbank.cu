#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "sharedandbank.h"

int** scannedSUMS;

namespace StreamCompaction {
	namespace SharedAndBank {
		using StreamCompaction::Common::PerformanceTimer;
		PerformanceTimer& timer() {
			static PerformanceTimer timer;
			return timer;
		}

		__global__ void kernZeroExcessLeaves(const int pow2roundedsize, const int orig_size, int* data) {
			const int index = blockIdx.x * blockDim.x + threadIdx.x;
			if (index >= pow2roundedsize || index < orig_size) return;
			data[index] = 0;
		}

#define AVOIDBANKCONFLICT 0
		__global__ void kernScan(const int shMemEntries, int* idata, int* SUMS) {
			extern __shared__ int temp[];
			const int thid_blk = threadIdx.x;
			const int thid_grid = blockIdx.x * blockDim.x + threadIdx.x;
			

#if AVOIDBANKCONFLICT == 1
			const int ai = thid_blk;
			const int bi = thid_blk + blockDim.x;
			const int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
			const int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
			const int ai_grid = blockIdx.x*shMemEntries + thid_blk;
			const int bi_grid = ai_grid + blockDim.x;
			temp[ai + bankOffsetA] = idata[ai_grid];
			temp[bi + bankOffsetB] = idata[bi_grid];
#else
			temp[2*thid_blk]   = idata[2*thid_grid];
			temp[2*thid_blk+1] = idata[2*thid_grid+1];
#endif

			//Scan upswep
			int offset = 1;
			for (int d = shMemEntries>>1; d > 0; d >>= 1) {//runs ilog2(shMemEntries) number of times
				__syncthreads();
				if (thid_blk < d) {//last iter offset is 64, lchild should be 63 and rchild 127
					int lchild = offset*(2*thid_blk+1)-1;
					int rchild = offset*(2*thid_blk+2)-1;
#if AVOIDBANKCONFLICT == 1
					lchild += CONFLICT_FREE_OFFSET(lchild);
					rchild += CONFLICT_FREE_OFFSET(rchild);
#endif
					temp[rchild] += temp[lchild];
				}
				offset <<= 1;
			}

			//intermediate step, copy the block sums to SUMS 
			int lastindex = shMemEntries - 1;
#if AVOIDBANKCONFLICT == 1
			lastindex += CONFLICT_FREE_OFFSET(lastindex);
#endif

			if (gridDim.x > 1 && 0 == thid_blk) { SUMS[blockIdx.x] = temp[lastindex]; }

			//zero last element for this block
			if (0 == thid_blk) { temp[lastindex] = 0; }

			//scan downswep
			for (int d = 1; d < shMemEntries; d <<= 1) {//runs same amount as downsweep
				offset >>= 1;
				__syncthreads();
				if (thid_blk < d) {
					int lchild = offset*(2*thid_blk+1)-1;
					int rchild = offset*(2*thid_blk+2)-1;
#if AVOIDBANKCONFLICT == 1
					lchild += CONFLICT_FREE_OFFSET(lchild);
					rchild += CONFLICT_FREE_OFFSET(rchild);
#endif
					int otherparent = temp[lchild];
					temp[lchild] = temp[rchild];
					temp[rchild] += otherparent;
				}
			}
			__syncthreads();

#if AVOIDBANKCONFLICT == 1
			idata[ai_grid] = temp[ai + bankOffsetA];
			idata[bi_grid] = temp[bi + bankOffsetB];
#else
			idata[2*thid_grid]   = temp[2*thid_blk];
			idata[2*thid_grid+1] = temp[2*thid_blk+1];
#endif
		}

		__global__ void kernAddBack(const int n, int* idata, const int* scannedSumsLevel) {
			__shared__ int scannedSumForThisBlock;
			if (threadIdx.x == 0) { scannedSumForThisBlock = scannedSumsLevel[blockIdx.x]; }
			const int thid_grid = blockIdx.x*blockDim.x + threadIdx.x;
			__syncthreads();
			idata[2*thid_grid] += scannedSumForThisBlock;//add running total of all prev elements before this block to this block
			idata[2*thid_grid+1] += scannedSumForThisBlock;
		}

		void recursiveScan(const int n, const int level, int *idata) {
			//printf("\ncalling recursiveScan with pow2size: %i level: %i\n", n, level);

			//generate params for the kernel
			const int shMemEntries = blockSize << 1;
			const int shMemSize = shMemEntries * sizeof(int);
			const int blocksThisLevel = (n + shMemEntries - 1) / shMemEntries;
			const int pow2BlocksThisLevel = 1 << ilog2ceil(blocksThisLevel);


			//check if we are at the last level
			//via how many blocks have on this level
			//if not keep recursing 
			if (pow2BlocksThisLevel > 1) {

				//1. scan up and down this level, if its the first recursion
				//then the thread blocks just scan sections of 
				//their corresponding global input data.
				//2. recursiveScan on the last entries in each block
				//(they get copied to an array stored at scannedSUMS[level]
				//for this level during the kernel call
				//3. call the addBack kernel so we can recursively 
				//add back the reduced sums of each block back up through
				//scannedSUMS levels and then finally back to the final 
				//result odata. Doing this will allow us to arrive at our 
				//final inclusive scanned result. pretty cool. 
				kernScan<<<pow2BlocksThisLevel,blockSize,shMemSize>>>(
					shMemEntries, idata, scannedSUMS[level]);

				recursiveScan(pow2BlocksThisLevel, level+1, scannedSUMS[level]); //if you need odata and idata, then its scannedSUMS[level], scannedSUMS[level]

				kernAddBack<<<pow2BlocksThisLevel,blockSize>>>(n, idata, scannedSUMS[level]);

			} else {
				//last level, 1 block to run, just call the kernel, the recursive call that we
				//are currently in is for the last level of scannedSUMS[]
				//After this we start popping the recursive stack,
				//adding the SUMS back up the through the scannedSUMS levels
				//and then into the final result odata of the first recursive call

				kernScan<<<pow2BlocksThisLevel,blockSize,shMemSize>>>(
					shMemEntries, idata, scannedSUMS[level]);
				//gpuErrchk(hipPeekAtLastError());
				//gpuErrchk(hipDeviceSynchronize());
			}
		}

		void scan(const int n, int *odata, const int *idata) {
			int* dev_idata;
			//int* dev_odata;

			const int pow2roundedsize = 1 << ilog2ceil(n);
			const int numbytes_pow2roundedsize = pow2roundedsize * sizeof(int);
			const int numbytes_copy = n * sizeof(int);

			/////////////////////////////////////////
			//// ALLOC AND COPY TO DEVICE MEMORY ////
			/////////////////////////////////////////
			hipMalloc((void**)&dev_idata, numbytes_pow2roundedsize);
			checkCUDAError("hipMalloc dev_data failed!");
			hipMemcpy(dev_idata, idata, numbytes_copy, hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy from idata to dev_data failed!");
			//hipMalloc((void**)&dev_odata, numbytes_pow2roundedsize);
			//checkCUDAError("hipMalloc dev_data failed!");
			//hipMemcpy(dev_odata, idata, numbytes_copy, hipMemcpyHostToDevice);
			//checkCUDAError("hipMemcpy from idata to dev_data failed!");

			timer().startGpuTimer();
			StreamCompaction::SharedAndBank::scanNoMalloc(n, dev_idata);
			timer().endGpuTimer();

			hipMemcpy(odata, dev_idata, numbytes_copy, hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy from dev_odata to odata failed!");
			//hipMemcpy(odata, dev_odata, numbytes_copy, hipMemcpyDeviceToHost);
			//checkCUDAError("hipMemcpy from dev_odata to odata failed!");
			//hipFree(dev_odata);
			//checkCUDAError("hipFree(dev_odata) failed!");
			hipFree(dev_idata);
			checkCUDAError("hipFree(dev_idata) failed!");
		}

		void scanNoMalloc(const int n, int *dev_idata) {
			const int pow2roundedsize = 1 << ilog2ceil(n);
			const int numbytes_pow2roundedsize = pow2roundedsize * sizeof(int);
			const int numbytes_copy = n * sizeof(int);
			/////////////////////////////////////////////////////////
			//// ALLOC scannedSUMS[] NEEDED FOR RECURSION PASSES ////
			/////////////////////////////////////////////////////////
			//alloc SUMS device memory pointer array as this scan
			//process needs to be recursive for arbitrary array sizes and arbitrary block sizes
			const int shMemEntries = blockSize << 1;
			int scannedSUMSTotalLevels;
			//int* scannedSUMSEntriesPerLevel;
			{
				int level = 0;
				int blocksThisLevel = (pow2roundedsize + shMemEntries - 1) / shMemEntries;
				while (blocksThisLevel > 1) {
					level++;
					blocksThisLevel = (blocksThisLevel + shMemEntries - 1) / shMemEntries;
				}
				scannedSUMS = (int**)malloc(level * sizeof(int*));
				scannedSUMSTotalLevels = level;
				//scannedSUMSEntriesPerLevel = new int[level];

				level = 0;
				blocksThisLevel = (pow2roundedsize + shMemEntries - 1) / shMemEntries;
				while (blocksThisLevel > 1) {
					const int pow2BlocksThisLevel = 1 << ilog2ceil(blocksThisLevel);
					//scannedSUMSEntriesPerLevel[level] = pow2BlocksThisLevel;
					hipMalloc((void**)&scannedSUMS[level++], pow2BlocksThisLevel * sizeof(int));
					checkCUDAError("hipMalloc scannedSUMS[level++] failed!");
					blocksThisLevel = (blocksThisLevel + shMemEntries - 1) / shMemEntries;
				}
			}

			/////////////////////////////
			//// 0 PAD FOR POW2 SIZE ////
			/////////////////////////////
			int gridDim = (pow2roundedsize + blockSize - 1) / blockSize;
			kernZeroExcessLeaves<<<gridDim, blockSize>>>(pow2roundedsize, n, dev_idata);
			//kernZeroExcessLeaves<<<gridDim, blockSize>>>(pow2roundedsize, n, dev_odata);

			////////////////////////
			//// RECURSIVE SCAN ////
			////////////////////////
			recursiveScan(pow2roundedsize, 0, dev_idata);

			///////////////////////
			//// COPY AND FREE ////
			///////////////////////
			{//free scannedSUMS related memory
				for (int i = 0; i < scannedSUMSTotalLevels; ++i) {
					hipFree(scannedSUMS[i]);
					checkCUDAError("hipFree(scannedSUMS[i]) failed!");
				}
				free(scannedSUMS);
			}
		}

		int compact(const int n, int *odata, const int *idata) {
			const int numbytes_copy = n * sizeof(int);
			const int pow2roundedsize = 1 << ilog2ceil(n);
			const int numbytes_pow2roundedsize = pow2roundedsize * sizeof(int);
			int* dev_idata;

			hipMalloc((void**)&dev_idata, numbytes_copy);
			checkCUDAError("hipMalloc dev_idata failed!");

			hipMemcpy(dev_idata, idata, numbytes_copy, hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy idata to dev_idata failed!");

            timer().startGpuTimer();
			const int size = StreamCompaction::SharedAndBank::compactNoMalloc(n, dev_idata);
            timer().endGpuTimer();
			
			hipMemcpy(odata, dev_idata, numbytes_copy, hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy dev_odata to odata failed!");

			hipFree(dev_idata);
			checkCUDAError("hipFree of dev_idata failed!");

			return size;
		}

		int compactNoMalloc(const int n, int *dev_idata) {
			const int numbytes_copy = n * sizeof(int);
			const int pow2roundedsize = 1 << ilog2ceil(n);
			const int numbytes_pow2roundedsize = pow2roundedsize * sizeof(int);
			int* dev_boolsThenIndices;

			hipMalloc((void**)&dev_boolsThenIndices, numbytes_pow2roundedsize);
			checkCUDAError("hipMalloc dev_bools failed!");

			int boolsLAST;
			hipMemcpyAsync(&boolsLAST, dev_idata + n - 1, sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy dev_bools to boolsLAST failed!");

			const int gridDim = (n + blockSize - 1) / blockSize;

			StreamCompaction::Common::kernMapToBoolean<<<gridDim, blockSize>>>(n, dev_boolsThenIndices, dev_idata);

			StreamCompaction::SharedAndBank::scanNoMalloc(pow2roundedsize, dev_boolsThenIndices);
			int indicesLAST;
			hipMemcpyAsync(&indicesLAST, dev_boolsThenIndices + n - 1, sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy dev_indices to indicesLAST failed!");

			StreamCompaction::Common::kernScatter<<<gridDim, blockSize>>>(n, dev_idata, dev_boolsThenIndices);

			//find the size of our new compacted array
			hipDeviceSynchronize();
			const int size = indicesLAST + (boolsLAST == 0 ? 0 : 1);

			//free everything
			hipFree(dev_boolsThenIndices);
			checkCUDAError("hipFree of dev_bools failed!");

			return size;
		}
	}
}