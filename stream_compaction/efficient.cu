#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include <src/sceneStructs.h>

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

#define blockSize 32
#define MAX_BLOCK_SIZE 16
#define checkCUDAErrorWithLine(msg) ((void)0) 
        //checkCUDAError(msg, __LINE__)
#define USE_CUDA_DEV_SYNC 0

        __global__ void upSweepIteration(int n, int *odata, const int offset, const int halfOffset) {
          int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
          int nodeIdx = (idx + 1) * offset - 1;
          if (nodeIdx < n) {
            odata[nodeIdx] = odata[nodeIdx] + odata[nodeIdx - halfOffset];
          }
        }

        __global__ void setRootToZero(int n, int *odata) {
          odata[n - 1] = 0;
        }

        __global__ void downSweepIteration(int n, int *odata, const int offset, const int halfOffset) {
          int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
          int nodeIdx = (idx + 1) * offset - 1;
          if (nodeIdx < n) {
            int originalNodeValue = odata[nodeIdx];
            odata[nodeIdx] = odata[nodeIdx] + odata[nodeIdx - halfOffset];
            odata[nodeIdx - halfOffset] = originalNodeValue;
          }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         * internalUse specifies whether this is used as a helper function,
         * for example, in compact. If so, it assumes idata and odata are in
         * device memory and does not use gpuTimer.
         */
        void scan(int n, int *odata, const int *idata, bool internalUse) {
          if (n == 1) {
            odata[0] = 0;
            return;
          }
          // TODO: handle n <= 2 ???
          // nearest power of two
          const int bufSize = 1 << ilog2ceil(n);
          
          int *dev_buf;
          if (internalUse) {
            dev_buf = odata;
          }
          else {
            hipMalloc((void**)&dev_buf, bufSize * sizeof(int));
            checkCUDAErrorWithLine("malloc dev_buf error!!!");

            if (n != bufSize) {
              hipMemset(dev_buf + n, 0, (bufSize - n) * sizeof(int));
              checkCUDAErrorWithLine("memset dev_buf to 0 error!!!");
            }

            hipMemcpy(dev_buf, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAErrorWithLine("memcpy dev_buf error!!!");
          }



          int halfOffset = 1;
          int numThreads = bufSize / 2;
          dim3 numBlocks(1);
          int threadsPerBlock;

          hipDeviceSynchronize();
          checkCUDAErrorWithLine("cuda sync error!!!");

          if (!internalUse) {
            timer().startGpuTimer();
          }
          // skip offset = n because we overwrite root's value anyway
          for (int offset = 2; offset < bufSize; offset *= 2) {
            if (numThreads > MAX_BLOCK_SIZE) {
              numBlocks.x = numThreads / MAX_BLOCK_SIZE;
              //numBlocks = dim3(numThreads / MAX_BLOCK_SIZE);
              threadsPerBlock = MAX_BLOCK_SIZE;
            }
            else {
              numBlocks.x = 1;
              //numBlocks = dim3(1);
              threadsPerBlock = numThreads;
            }
            upSweepIteration<<<numBlocks, threadsPerBlock>>>(bufSize, dev_buf, offset, halfOffset);
            checkCUDAErrorWithLine("upSweep error!!!");
            halfOffset = offset;
            numThreads /= 2;
          }

          setRootToZero << <dim3(1), 1 >> > (bufSize, dev_buf);

          int offset = bufSize;
          numThreads = 1;
          for (int halfOffset = bufSize / 2; halfOffset >= 1; halfOffset /= 2) {
            if (numThreads > MAX_BLOCK_SIZE) {
              numBlocks.x = numThreads / MAX_BLOCK_SIZE;
              //numBlocks = dim3(numThreads / MAX_BLOCK_SIZE);
              threadsPerBlock = MAX_BLOCK_SIZE;
            }
            else {
              numBlocks.x = 1;
              //numBlocks = dim3(1);
              threadsPerBlock = numThreads;
            }
            downSweepIteration << <numBlocks, threadsPerBlock >> >(bufSize, dev_buf, offset, halfOffset);
            checkCUDAErrorWithLine("downSweep error!!!");
            offset = halfOffset;
            numThreads *= 2;
          }

          if (!internalUse) {
            timer().endGpuTimer();
            hipMemcpy(odata, dev_buf, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAErrorWithLine("memcpy dev_buf to host error!!!");
            hipFree(dev_buf);
            checkCUDAErrorWithLine("free dev_buf error!!!");
          }
      
        }

        __global__ void map(int n, int *odata, const int *idata) {
          int idx = (blockIdx.x * blockDim.x) + threadIdx.x; // TODO?
          if (idx < n) {
            odata[idx] = (idata[idx] != 0) ? 1 : 0;
          }
        }

        __global__ void scatter(int n, int *odata, const int *postMapData, const int *postScanData, const int *originalData) {
          int idx = (blockIdx.x * blockDim.x) + threadIdx.x; // TODO?
          if (idx < n && postMapData[idx]) {
            odata[postScanData[idx]] = originalData[idx];
          }
        }

        __global__ void getCompactedSize(int n, int *odata, const int *postMapData, const int *postScanData) {
          *odata = postScanData[n - 1] + (postMapData[n - 1] ? 1 : 0);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
          int *dev_originalData;
          int *dev_postMapBuf;
          int *dev_postScanBuf;
          int *dev_postScatterBuf;
          int *dev_scatteredSize;

          hipMalloc((void**)&dev_originalData, n * sizeof(int));
          checkCUDAErrorWithLine("malloc dev_originalData error!!!");

          hipMalloc((void**)&dev_postMapBuf, n * sizeof(int));
          checkCUDAErrorWithLine("malloc dev_postMapBuf error!!!");

          // needs to be power of 2 for scan to work
          const int postScanBufSize = 1 << ilog2ceil(n);

          hipMalloc((void**)&dev_postScanBuf, postScanBufSize * sizeof(int));
          checkCUDAErrorWithLine("malloc dev_postScanBuf error!!!");

          if (postScanBufSize != n) {
            hipMemset(dev_postScanBuf, 0, postScanBufSize * sizeof(int));
            checkCUDAErrorWithLine("memset dev_postScanBuf to 0 error!!!");
          }

          hipMalloc((void**)&dev_postScatterBuf, n * sizeof(int));
          checkCUDAErrorWithLine("malloc dev_postScatterBuf error!!!");

          hipMalloc((void**)&dev_scatteredSize, sizeof(int));
          checkCUDAErrorWithLine("malloc dev_scatteredSize error!!!");

          hipMemcpy(dev_originalData, idata, n * sizeof(int), hipMemcpyHostToDevice);
          checkCUDAErrorWithLine("memcpy dev_originalData from host error!!!");

          dim3 numBlocks((n + blockSize - 1) / blockSize);

          timer().startGpuTimer();

          map<<<numBlocks, blockSize>>>(n, dev_postMapBuf, dev_originalData);
          checkCUDAErrorWithLine("map error!!!");

          hipMemcpy(dev_postScanBuf, dev_postMapBuf, n * sizeof(int), hipMemcpyDeviceToDevice);
          checkCUDAErrorWithLine("memcpy map to scan error!!!");

          scan(n, dev_postScanBuf, dev_postMapBuf, true);
          checkCUDAErrorWithLine("scan error!!!");
          scatter << <numBlocks, blockSize >> > (n, dev_postScatterBuf, dev_postMapBuf, dev_postScanBuf, dev_originalData);
          checkCUDAErrorWithLine("scatter error!!!");
          getCompactedSize<<<dim3(1), 1>>>(n, dev_scatteredSize, dev_postMapBuf, dev_postScanBuf);
          checkCUDAErrorWithLine("get size error!!!");
          
          timer().endGpuTimer();

          int scatteredSize;

          hipMemcpy(&scatteredSize, dev_scatteredSize, sizeof(int), hipMemcpyDeviceToHost);
          checkCUDAErrorWithLine("memcpy dev_scatteredSize to host error!!!");

          hipMemcpy(odata, dev_postScatterBuf, scatteredSize * sizeof(int), hipMemcpyDeviceToHost);
          checkCUDAErrorWithLine("memcpy dev_postScatterBuf to host error!!!");

          hipFree(dev_originalData);
          checkCUDAErrorWithLine("free dev_originalData error!!!");

          hipFree(dev_postMapBuf);
          checkCUDAErrorWithLine("free dev_postMapBuf error!!!");

          hipFree(dev_postScanBuf);
          checkCUDAErrorWithLine("free dev_postScanBuf error!!!");

          hipFree(dev_postScatterBuf);
          checkCUDAErrorWithLine("free dev_postScatterBuf error!!!");

          hipFree(dev_scatteredSize);
          checkCUDAErrorWithLine("free dev_scatteredSize error!!!");

          return scatteredSize;
        }
    }
}
