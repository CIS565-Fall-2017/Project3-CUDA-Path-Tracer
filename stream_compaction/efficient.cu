#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define blocksize 128

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		__global__ void upSweep(int n, int pow2dPlus1, int pow2d, int *odata, bool reachedRoot)
		{
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= n) {
				return;
			}

			if (reachedRoot) {
				odata[n - 1] = 0;
			}
			else {
				index *= pow2dPlus1;
				if (index < n)
					odata[index + pow2dPlus1 - 1] += odata[index + pow2d - 1];
			}
		}

		__global__ void downSweep(int n, int pow2dPlus1, int pow2d, int *odata)
		{
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= n) {
				return;
			}

			index *= pow2dPlus1;
			if (index < n) {
				int t = odata[index + pow2d - 1];
				odata[index + pow2d - 1] = odata[index + pow2dPlus1 - 1];
				odata[index + pow2dPlus1 - 1] += t;
			}
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
			dim3 fullBlocksPerGrid((n + blocksize - 1) / blocksize);
			
			// Get the next power of 2
			int currPow = ilog2ceil(n) - 1;
			int nextPow = 2 << currPow;

			int *temp = new int[nextPow];
			for (int i = 0; i < nextPow; i++) {
				if (i < n) {
					temp[i] = idata[i];
				}
				// Fill the rest of the array with 0 if not a power of 2.
				else {
					temp[i] = 0;
				}
			}

			int *out;
			hipMalloc((void**)&out, nextPow * sizeof(int));
			checkCUDAError("hipMalloc out failed!");
			hipMemcpy(out, temp, sizeof(int) * nextPow, hipMemcpyHostToDevice);

			timer().startGpuTimer();
            // TODO

			// Up-Sweep
			for (int d = 0; d <= ilog2ceil(nextPow) - 1; d++) {
				int pow2dPlus1 = pow(2, d + 1);
				int pow2d = pow(2, d);

				// If we hit the end of the depth then we should be writing to the very last spot in the array.
				bool reachedRoot = (d == ilog2ceil(nextPow) - 1);
				upSweep << < fullBlocksPerGrid, blocksize >> > (nextPow, pow2dPlus1, pow2d, out, reachedRoot);
			}

			// Down-Sweep
			for (int d = ilog2ceil(nextPow) - 1; d >= 0; d--) {
				int pow2dPlus1 = pow(2, d + 1);
				int pow2d = pow(2, d);
			
				downSweep << < fullBlocksPerGrid, blocksize >> > (nextPow, pow2dPlus1, pow2d, out);
			}

            timer().endGpuTimer();

			// Copy final values into odata
			hipMemcpy(odata, out, sizeof(int) * nextPow, hipMemcpyDeviceToHost);

			delete[]temp;
			hipFree(out);
        }

		void exclusiveScan(int n, int *odata, const int *idata) {
			dim3 fullBlocksPerGrid((n + blocksize - 1) / blocksize);

			// Get the next power of 2
			int currPow = ilog2ceil(n) - 1;
			int nextPow = 2 << currPow;

			int *temp = new int[nextPow];
			for (int i = 0; i < nextPow; i++) {
				if (i < n) {
					temp[i] = idata[i];
				}
				// Fill the rest of the array with 0 if not a power of 2.
				else {
					temp[i] = 0;
				}
			}

			int *out;
			hipMalloc((void**)&out, nextPow * sizeof(int));
			checkCUDAError("hipMalloc out failed!");
			hipMemcpy(out, temp, sizeof(int) * nextPow, hipMemcpyHostToDevice);

			// Up-Sweep
			for (int d = 0; d <= ilog2ceil(nextPow) - 1; d++) {
				//int pow2dPlus1 = pow(2, d + 1);
				//int pow2d = pow(2, d);

				int pow2dPlus1 = pow(2, d + 1);
				int pow2d = pow(2, d);

				// If we hit the end of the depth then we should be writing to the very last spot in the array.
				bool reachedRoot = (d == ilog2ceil(nextPow) - 1);
				upSweep << < fullBlocksPerGrid, blocksize >> > (nextPow, pow2dPlus1, pow2d, out, reachedRoot);
			}

			// Down-Sweep
			for (int d = ilog2ceil(nextPow) - 1; d >= 0; d--) {
				int pow2dPlus1 = pow(2, d + 1);
				int pow2d = pow(2, d);

				downSweep << < fullBlocksPerGrid, blocksize >> > (nextPow, pow2dPlus1, pow2d, out);
			}

			// Copy final values into odata
			hipMemcpy(odata, out, sizeof(int) * nextPow, hipMemcpyDeviceToHost);

			delete[]temp;
			hipFree(out);
		}


        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
		int compact(int n, int *odata, const int *idata) {
			dim3 fullBlocksPerGrid((n + blocksize - 1) / blocksize);

			// Get the next power of 2
			int currPow = ilog2ceil(n) - 1;
			int nextPow = 2 << currPow;

			int *dev_in;
			hipMalloc((void**)&dev_in, sizeof(int) * nextPow);
			hipMemcpy(dev_in, idata, sizeof(int) * nextPow, hipMemcpyHostToDevice);

			int *dev_out;
			hipMalloc((void**)&dev_out, sizeof(int) * nextPow);

			int *dev_indices;
			hipMalloc((void**)&dev_indices, sizeof(int) * nextPow);

			int *bools = new int[nextPow];
			int *dev_bools;
			hipMalloc((void**)&dev_bools, sizeof(int) * nextPow);

			timer().startGpuTimer();
			// TODO

			StreamCompaction::Common::kernMapToBoolean << < fullBlocksPerGrid, blocksize >> > (nextPow, dev_bools, dev_in);
			hipMemcpy(bools, dev_bools, sizeof(int) * nextPow, hipMemcpyDeviceToHost);

			exclusiveScan(nextPow, dev_indices, bools);

			StreamCompaction::Common::kernScatter << < fullBlocksPerGrid, blocksize >> > (nextPow, dev_out, dev_in, dev_bools, dev_indices);

			timer().endGpuTimer();

			int numElements = 0;
			for (int i = 0; i < n; i++) {
				if (bools[i] == 1) {
					numElements++;
				}
			}

			hipMemcpy(odata, dev_out, sizeof(int) * numElements, hipMemcpyDeviceToHost);

			hipFree(dev_in);
			hipFree(dev_out);
			hipFree(dev_indices);
			hipFree(dev_bools);

			delete[]bools;

			return numElements;
		}
    }
}
