#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		__global__ void upSweep(int n, int factorPlusOne, int factor, int addTimes, int *idata)
		{
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;

			if (index < addTimes)
			{
				int newIndex = (factorPlusOne * (index + 1)) - 1;

				if (newIndex < n)
				{
					idata[newIndex] += idata[newIndex - factor];

					//if (newIndex == n - 1)
					//{
					//	idata[newIndex] = 0;
					//}
				}
			}

			

		}//end upSweep function

		__global__ void downSweep(int n, int factorPlusOne, int factor, int addTimes, int *idata)
		{
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;

			if (index < addTimes)
			{
				int newIndex = (factorPlusOne * (index + 1)) - 1;

				if (newIndex < n)
				{
					int leftChild = idata[newIndex - factor];
					idata[newIndex - factor] = idata[newIndex];
					idata[newIndex] += leftChild;
				}
			}

		}//end downSweep function


		__global__ void resizeArray(int n, int new_n, int *idata)
		{
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;
			if (index < new_n && index >= n)
			{
				idata[index] = 0;
			}
		}


        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.

		 * Notes:
			 Most of the text in Part 2 applies.
			 This uses the "Work-Efficient" algorithm from GPU Gems 3, Section 39.2.2.
			 This can be done in place - it doesn't suffer from the race conditions of the naive method, 
			 since there won't be a case where one thread writes to and another thread reads from the same location in the array.
			 Beware of errors in Example 39-2. Test non-power-of-two-sized arrays.
			 Since the work-efficient scan operates on a binary tree structure, it works best with arrays with power-of-two length. 
			 Make sure your implementation works on non-power-of-two sized arrays (see ilog2ceil). 
			 This requires extra memory, so your intermediate array sizes 
			 will need to be rounded to the next power of two.
         */
        void scan(int n, int *odata, const int *idata) {
			// TODO
			
			//If non-power-of-two sized array, round to next power of two
			int new_n = 1 << ilog2ceil(n);

			dim3 fullBlocksPerGrid((new_n + blockSize - 1) / blockSize);

			int *inArray;
			hipMalloc((void**)&inArray, new_n * sizeof(int));
			checkCUDAError("hipMalloc inArray failed!");

			//Copy input data to device array and resize if necessary
			hipMemcpy(inArray, idata, sizeof(int) * new_n, hipMemcpyHostToDevice);
			resizeArray<<<fullBlocksPerGrid, blockSize>>>(n, new_n, inArray);

			bool timerHasStartedElsewhere = false;
			try
			{
				timer().startGpuTimer();
			}
			catch (std::runtime_error &e)
			{
				timerHasStartedElsewhere = true;
			}

			dim3 newNumBlocks = fullBlocksPerGrid;
			

			//Up sweep
			for (int d = 0; d <= ilog2ceil(n) - 1; d++)
			{
				int factorPlusOne = 1 << (d + 1);	//2^(d + 1)
				int factor = 1 << d;				//2^d

				int addTimes = 1 << (ilog2ceil(n) - 1 - d);
				
				newNumBlocks = ((new_n / factorPlusOne) + blockSize - 1) / blockSize;

				upSweep<<<newNumBlocks, blockSize>>>(new_n, factorPlusOne, factor, addTimes, inArray);

				//Make sure the GPU finishes before the next iteration of the loop
				hipDeviceSynchronize();

				
			}

			//Down sweep
			int lastElem = 0;
			hipMemcpy(inArray + (new_n - 1), &lastElem, sizeof(int) * 1, hipMemcpyHostToDevice);

			for (int d = ilog2ceil(n) - 1; d >= 0; d--)
			{
				int factorPlusOne = 1 << (d + 1);	//2^(d + 1)
				int factor = 1 << d;				//2^d

				int addTimes = 1 << (ilog2ceil(n) - 1 - d);

				newNumBlocks = ((new_n / factor) + blockSize - 1) / blockSize;

				downSweep<<<newNumBlocks, blockSize>>>(new_n, factorPlusOne, factor, addTimes, inArray);


				hipDeviceSynchronize();
			}

			if (!timerHasStartedElsewhere)
			{
				timer().endGpuTimer();
			}

			//Transfer to odata
			hipMemcpy(odata, inArray, sizeof(int) * (new_n), hipMemcpyDeviceToHost);

			//Free the arrays
			hipFree(inArray);
        }//end scan function 

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
			// TODO
			
			int new_n = 1 << ilog2ceil(n);

			dim3 fullBlocksPerGrid((new_n + blockSize - 1) / blockSize);
			
			int *inArray;
			int *boolsArray;
			hipMalloc((void**)&inArray, new_n * sizeof(int));
			checkCUDAError("hipMalloc inArray failed!");
			hipMalloc((void**)&boolsArray, new_n * sizeof(int));
			checkCUDAError("hipMalloc boolsArray failed!");

			int* scan_in = (int *)malloc(sizeof(int) * new_n);
			int* scan_out = (int *)malloc(sizeof(int) * new_n);

			int *scatter_in;
			int *scatter_out;
			hipMalloc((void**)&scatter_in, new_n * sizeof(int));
			checkCUDAError("hipMalloc scatter_in failed!");
			hipMalloc((void**)&scatter_out, new_n * sizeof(int));
			checkCUDAError("hipMalloc scatter_out failed!");

			hipDeviceSynchronize();


			//Copy input data to device array
			hipMemcpy(inArray, idata, sizeof(int) * new_n, hipMemcpyHostToDevice);
			resizeArray<<<fullBlocksPerGrid, blockSize>>>(n, new_n, inArray);

			timer().startGpuTimer();

			//Call kernMapToBoolean to map values to bool array
			Common::kernMapToBoolean<<<fullBlocksPerGrid, blockSize>>>(new_n, boolsArray, inArray);

			//Copy back to host array, find how many fulfilled condition, and run exclusive scan
			hipMemcpy(scan_in, boolsArray, sizeof(int) * new_n, hipMemcpyDeviceToHost);

			int numPassedElements = 0;
			for (int i = 0; i < new_n; i++)
			{
				if (scan_in[i] == 1)
				{
					numPassedElements++;
				}
			}

			scan(new_n, scan_out, scan_in);

			//Copy output of CPU scan to scatter device array
			hipMemcpy(scatter_in, scan_out, sizeof(int) * new_n, hipMemcpyHostToDevice);
			
			//Call kernScatter with scanned boolsArray
			Common::kernScatter<<<fullBlocksPerGrid, blockSize>>>(new_n, scatter_out, inArray, boolsArray, scatter_in);

			timer().endGpuTimer();

			//SCATTER OUT ISNT GONNA BE THE SAME SIZE AS N
			//Should I replace n with numPassedElements? 
			hipMemcpy(odata, scatter_out, sizeof(int) * numPassedElements, hipMemcpyDeviceToHost);

			//Free the arrays
			free(scan_in);
			free(scan_out);
			hipFree(inArray);
			hipFree(boolsArray);
			hipFree(scatter_in);
			hipFree(scatter_out);
			checkCUDAError("hipFree failed!");

            return numPassedElements;

        }//end compact function
    }//end namespace Efficient
}//end namespace StreamCompaction
