#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

//static int usetimer = 1;
namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		__global__ void kernScanUp(const int pow2roundedsize, const int indexscaling, const int offset, int* data) {
			//shift orig index up by 1 (otherwise thread 0 wouldn't pick up the index modifications), scale it, then shift back down
			const int index = (indexscaling * (blockIdx.x * blockDim.x + threadIdx.x + 1)) - 1;
			if (index >= pow2roundedsize) return;
			data[index] += data[index - offset];
		}

		__global__ void kernScanDown(const int pow2roundedsize, const int indexscaling, const int offset, int* data) {
			//shift orig index up by 1 (otherwise thread 0 wouldn't pick up the index modifications), scale it, then shift back down
			const int index = (indexscaling * (blockIdx.x * blockDim.x + threadIdx.x + 1)) - 1;
			if (index >= pow2roundedsize) return;
			int oldparent = data[index];
			data[index] += data[index - offset];
			data[index - offset] = oldparent;
		}

		__global__ void kernZeroExcessLeaves(const int pow2roundedsize, const int orig_size, int* data) {
			const int index = blockIdx.x * blockDim.x + threadIdx.x;
			if (index >= pow2roundedsize || index < orig_size) return;
			data[index] = 0;
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(const int n, int *odata, const int *idata) {
			int* dev_data;
			const int pow2roundedsize = 1 << ilog2ceil(n);
			const int numbytes_pow2roundedsize = pow2roundedsize * sizeof(int);
			const int numbytes_copy = n * sizeof(int);

			hipMalloc((void**)&dev_data, numbytes_pow2roundedsize);
			checkCUDAError("hipMalloc dev_data failed!");

			hipMemcpy(dev_data, idata, numbytes_copy, hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy from idata to dev_data failed!");

			int gridDim = (pow2roundedsize + blockSize - 1) / blockSize;

			//the algo works on pow2 sized arrays so we size up the array to the next pow 2 if it wasn't a pow of 2 to begin with
			//then we need to fill data after index n-1 with zeros 
			kernZeroExcessLeaves<<<gridDim, blockSize>>>(pow2roundedsize, n, dev_data);

			//if (usetimer) { timer().startGpuTimer(); }
			timer().startGpuTimer();
			for (int offset = 1; offset < pow2roundedsize; offset <<= 1) {
				gridDim = ((pow2roundedsize >> ilog2(offset << 1)) + blockSize - 1) / blockSize;
				//this fails when blocksize is more than 128
				kernScanUp<<<gridDim, blockSize>>>(pow2roundedsize, offset << 1, offset, dev_data);
			}

			//make sure last index value is 0 before we downsweep
			const int zero = 0;
			hipMemcpy(dev_data + pow2roundedsize - 1, &zero, sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy from zero to dev_data failed!");

			for (int offset = pow2roundedsize >> 1; offset > 0; offset >>= 1) {
				gridDim = ((pow2roundedsize >> ilog2(offset << 1)) + blockSize - 1) / blockSize;
				kernScanDown<<<gridDim, blockSize>>>(pow2roundedsize, offset << 1, offset, dev_data);
			}
			timer().endGpuTimer();

			hipMemcpy(odata, dev_data, numbytes_copy, hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy from dev_data to odata failed!");

			hipFree(dev_data);
			checkCUDAError("hipFree(dev_data) failed!");
        }


        void scan_notimer(const int n, int *odata, const int *idata) {
			int* dev_data;
			const int pow2roundedsize = 1 << ilog2ceil(n);
			const int numbytes_pow2roundedsize = pow2roundedsize * sizeof(int);
			const int numbytes_copy = n * sizeof(int);

			hipMalloc((void**)&dev_data, numbytes_pow2roundedsize);
			checkCUDAError("hipMalloc dev_data failed!");

			hipMemcpy(dev_data, idata, numbytes_copy, hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy from idata to dev_data failed!");

			int gridDim = (pow2roundedsize + blockSize - 1) / blockSize;

			//the algo works on pow2 sized arrays so we size up the array to the next pow 2 if it wasn't a pow of 2 to begin with
			//then we need to fill data after index n-1 with zeros 
			kernZeroExcessLeaves<<<gridDim, blockSize>>>(pow2roundedsize, n, dev_data);

			for (int offset = 1; offset < pow2roundedsize; offset <<= 1) {
				gridDim = ((pow2roundedsize >> ilog2(offset<<1)) + blockSize - 1) / blockSize;
				kernScanUp<<<gridDim, blockSize>>>(pow2roundedsize, offset << 1, offset, dev_data);
			}

			//make sure last index value is 0 before we downsweep
			const int zero = 0;
			hipMemcpy(dev_data + pow2roundedsize - 1, &zero, sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy from zero to dev_data failed!");

			for (int offset = pow2roundedsize >> 1; offset > 0; offset >>= 1) {
				gridDim = ((pow2roundedsize >> ilog2(offset<<1)) + blockSize - 1) / blockSize;
				kernScanDown<<<gridDim, blockSize>>>(pow2roundedsize, offset << 1, offset, dev_data);
			}

			hipMemcpy(odata, dev_data, numbytes_copy, hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy from dev_data to odata failed!");

			hipFree(dev_data);
			checkCUDAError("hipFree(dev_data) failed!");
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(const int n, int *odata, const int *idata) {
			const int numbytes_copy = n * sizeof(int);
			const int pow2roundedsize = 1 << ilog2ceil(n);
			const int numbytes_pow2roundedsize = pow2roundedsize * sizeof(int);
			int* dev_idata;
			int* dev_odata;
			int* dev_bools;
			int* dev_indices;


			hipMalloc((void**)&dev_idata, numbytes_copy);
			checkCUDAError("hipMalloc dev_idata failed!");

			hipMalloc((void**)&dev_odata, numbytes_copy);
			checkCUDAError("hipMalloc dev_odata failed!");

			hipMalloc((void**)&dev_bools, numbytes_copy);
			checkCUDAError("hipMalloc dev_bools failed!");

			hipMalloc((void**)&dev_indices, numbytes_pow2roundedsize);
			checkCUDAError("hipMalloc dev_indices failed!");

			hipMemcpy(dev_idata, idata, numbytes_copy, hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy idata to dev_idata failed!");

			const int gridDim = (n + blockSize - 1) / blockSize;

            timer().startGpuTimer();

			StreamCompaction::Common::kernMapToBoolean<<<gridDim, blockSize>>>(n, dev_bools, dev_idata);

			{//generate scan
				hipMemcpy(dev_indices, dev_bools, numbytes_copy, hipMemcpyDeviceToDevice);
				checkCUDAError("hipMemcpy from to dev_bools to dev_indices failed!");

				int gridDimScan = (pow2roundedsize + blockSize - 1) / blockSize;

				//the algo works on pow2 sized arrays so we size up the array to the next pow 2 if it wasn't a pow of 2 to begin with
				//then we need to fill data after index n-1 with zeros 
				StreamCompaction::Efficient::kernZeroExcessLeaves << <gridDimScan, blockSize >> > (pow2roundedsize, n, dev_indices);

				for (int offset = 1; offset < pow2roundedsize; offset <<= 1) {
					gridDimScan = ((pow2roundedsize >> ilog2(offset << 1)) + blockSize - 1) / blockSize;
					StreamCompaction::Efficient::kernScanUp << <gridDimScan, blockSize >> > (pow2roundedsize, offset << 1, offset, dev_indices);
				}

				//make sure last index value is 0 before we downsweep
				const int zero = 0;
				hipMemcpy(dev_indices + pow2roundedsize - 1, &zero, sizeof(int), hipMemcpyHostToDevice);
				checkCUDAError("hipMemcpy from zero to dev_data failed!");

				for (int offset = pow2roundedsize >> 1; offset > 0; offset >>= 1) {
					gridDimScan = ((pow2roundedsize >> ilog2(offset << 1)) + blockSize - 1) / blockSize;
					StreamCompaction::Efficient::kernScanDown << <gridDimScan, blockSize >> > (pow2roundedsize, offset << 1, offset, dev_indices);
				}
			}//end generate scan

			//StreamCompaction::Common::kernScatter<<<gridDim, blockSize>>>(n, dev_odata, dev_idata, dev_bools, dev_indices);
			//StreamCompaction::Common::kernScatter<<<gridDim, blockSize>>>(n, dev_odata, dev_idata, dev_indices);
			StreamCompaction::Common::kernScatter<<<gridDim, blockSize>>>(n, dev_idata, dev_indices);

            timer().endGpuTimer();
			
			//hipMemcpy(odata, dev_odata, numbytes_copy, hipMemcpyDeviceToHost);
			hipMemcpy(odata, dev_idata, numbytes_copy, hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy dev_odata to odata failed!");

			int indicesLAST;
			hipMemcpy(&indicesLAST, dev_indices + n - 1, sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy dev_indices to indicesLAST failed!");
			int boolsLAST;
			hipMemcpy(&boolsLAST, dev_bools + n - 1, sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy dev_bools to boolsLAST failed!");
			const int size = indicesLAST + boolsLAST;

			hipFree(dev_idata);
			checkCUDAError("hipFree of dev_idata failed!");

			hipFree(dev_odata);
			checkCUDAError("hipFree of dev_odata failed!");
			
			hipFree(dev_bools);
			checkCUDAError("hipFree of dev_bools failed!");

			hipFree(dev_indices);
			checkCUDAError("hipFree of dev_indices failed!");

			return size;
        }
    }
}
