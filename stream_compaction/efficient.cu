#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define blockSize 128

namespace StreamCompaction {
  namespace Efficient {
    using StreamCompaction::Common::PerformanceTimer;
    PerformanceTimer& timer()
    {
      static PerformanceTimer timer;
      return timer;
    }

    __global__ void kernUpSweep(int n, int* idata, int shift) {
      int index = threadIdx.x + (blockIdx.x * blockDim.x);
      if (index >= n) {
        return;
      }

      int offset = (shift << 1);
      if (index % offset == 0 && index + offset <= n) {
        idata[index + offset - 1] += idata[index + shift - 1];
      }
    }

    __global__ void kernDownSweep(int n, int* idata, int shift) {
      int index = threadIdx.x + (blockIdx.x * blockDim.x);
      if (index >= n) {
        return;
      }

      int offset = (shift << 1);
      if (index % offset == 0 && index + offset <= n) {
        int temp = idata[index + shift - 1];
        idata[index + shift - 1] = idata[index + offset - 1];
        idata[index + offset - 1] += temp;
      }
    }

    /**
      * Performs prefix-sum (aka scan) on idata, storing the result into odata.
      */
    void scan(int n, int *odata, const int *idata) {
      int maxN = (1 << ilog2ceil(n));
      dim3 fullBlocksPerGrid((maxN + blockSize - 1) / blockSize);

      int* idataSwap;

      hipMalloc((void**)&idataSwap, maxN * sizeof(int));
      checkCUDAError("hipMalloc for idataSwap failed");

      hipMemset(idataSwap, 0, maxN * sizeof(int));
      checkCUDAError("hipMemset for idataSwap failed");

      // Copy from CPU to GPU
      hipMemcpy(idataSwap, idata, n * sizeof(int), hipMemcpyHostToDevice);
      checkCUDAError("hipMemcpy for idataSwap failed");

      timer().startGpuTimer();

      // Up-sweep
      for (int depth = 0; depth < ilog2ceil(n); depth++) {
        int shift = (1 << depth);

        kernUpSweep << <fullBlocksPerGrid, blockSize >> >(maxN, idataSwap, shift);
        checkCUDAError("kernUpSweep failed");
      }

      hipMemset(idataSwap + maxN - 1, 0, sizeof(int));
        
      // Down-sweep
      for (int depth = ilog2ceil(n) - 1; depth >= 0; depth--) {
        int shift = (1 << depth);

        kernDownSweep << <fullBlocksPerGrid, blockSize >> >(maxN, idataSwap, shift);
        checkCUDAError("kernUpSweep failed");
      }

      timer().endGpuTimer();

      // Copy from GPU back to CPU
      hipMemcpy(odata, idataSwap, n * sizeof(int), hipMemcpyDeviceToHost);
      checkCUDAError("hipMemcpy for idataSwap failed");

      hipFree(idataSwap);
    }

    /**
      * Performs stream compaction on idata, storing the result into odata.
      * All zeroes are discarded.
      *
      * @param n      The number of elements in idata.
      * @param odata  The array into which to store elements.
      * @param idata  The array of elements to compact.
      * @returns      The number of elements remaining after compaction.
      */
    int compact(int n, int *odata, const int *idata) {
        
      dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

      // Allocate extra buffers
      int* odataSwap;
      hipMalloc((void**)&odataSwap, n * sizeof(int));
      checkCUDAError("hipMalloc for odataSwap failed");

      int* idataSwap;
      hipMalloc((void**)&idataSwap, n * sizeof(int));
      checkCUDAError("hipMalloc for idataSwap failed");

      int* boolsArr;
      hipMalloc((void**)&boolsArr, n * sizeof(int));
      checkCUDAError("hipMalloc for boolsArr failed");

      int* indicesArr;
      hipMalloc((void**)&indicesArr, n * sizeof(int));
      checkCUDAError("hipMalloc for scan_result failed");

      // Copy from CPU to GPU
      hipMemcpy(odataSwap, odata, n * sizeof(int), hipMemcpyHostToDevice);
      checkCUDAError("hipMemcpy for odataSwap failed");

      hipMemcpy(idataSwap, idata, n * sizeof(int), hipMemcpyHostToDevice);
      checkCUDAError("hipMemcpy for idataSwap failed");

      timer().startGpuTimer();

      // Map input array to a temp array of 0s and 1s
      StreamCompaction::Common::kernMapToBoolean << <fullBlocksPerGrid, blockSize >> >(n, boolsArr, idataSwap);
      checkCUDAError("kernMapToBoolean failed");

      // Scan
      scan(n, indicesArr, boolsArr);

      // Scatter
      StreamCompaction::Common::kernScatter << <fullBlocksPerGrid, blockSize >> >(n, odataSwap, idataSwap, boolsArr, indicesArr);
      checkCUDAError("kernScatter failed");

      timer().endGpuTimer();

      // Copy over compacted data from GPU to CPU
      hipMemcpy(odata, odataSwap, n * sizeof(int), hipMemcpyDeviceToHost);
      checkCUDAError("hipMemcpy for odataSwap failed");

		  // Grab remaining number of elements
		  int remainingNBools = 0;
		  hipMemcpy(&remainingNBools, boolsArr + n - 1, sizeof(int), hipMemcpyDeviceToHost);

		  int remainingNIndices = 0;
		  hipMemcpy(&remainingNIndices, indicesArr + n - 1, sizeof(int), hipMemcpyDeviceToHost);
	
      hipFree(odataSwap);
      hipFree(idataSwap);
      hipFree(boolsArr);
      hipFree(indicesArr);
        
      return remainingNBools + remainingNIndices;
    }

    __global__ void kernMapToBooleanPaths(int n, int *bools, const PathSegment *idata) {
      int index = threadIdx.x + (blockIdx.x * blockDim.x);
      if (index >= n) {
        return;
      }

      if (idata[index].remainingBounces != 0) {
        bools[index] = 1;
      }
      else {
        bools[index] = 0;
      }
    }

    __global__ void kernScatterPaths(int n, PathSegment *odata,
      const PathSegment *idata, const int *bools, const int *indices) {
      int index = threadIdx.x + (blockIdx.x * blockDim.x);
      if (index >= n) {
        return;
      }

      if (bools[index] == 1) {
        odata[indices[index]] = idata[index];
      }
    }

    void scanPaths(int n, int *odata, const int *idata) {
      int maxN = (1 << ilog2ceil(n));
      dim3 fullBlocksPerGrid((maxN + blockSize - 1) / blockSize);

      int* idataSwap;

      hipMalloc((void**)&idataSwap, maxN * sizeof(int));
      checkCUDAError("hipMalloc for idataSwap failed");

      hipMemset(idataSwap, 0, maxN * sizeof(int));
      checkCUDAError("hipMemset for idataSwap failed");

      // Copy from GPU to GPU
      hipMemcpy(idataSwap, idata, n * sizeof(int), hipMemcpyDeviceToDevice);
      checkCUDAError("hipMemcpy for idataSwap failed");


      // Up-sweep
      for (int depth = 0; depth < ilog2ceil(n); depth++) {
        int shift = (1 << depth);

        kernUpSweep << <fullBlocksPerGrid, blockSize >> >(maxN, idataSwap, shift);
        checkCUDAError("kernUpSweep failed");
      }

      hipMemset(idataSwap + maxN - 1, 0, sizeof(int));

      // Down-sweep
      for (int depth = ilog2ceil(n) - 1; depth >= 0; depth--) {
        int shift = (1 << depth);

        kernDownSweep << <fullBlocksPerGrid, blockSize >> >(maxN, idataSwap, shift);
        checkCUDAError("kernUpSweep failed");
      }


      // Copy from GPU back to GPU
      hipMemcpy(odata, idataSwap, n * sizeof(int), hipMemcpyDeviceToDevice);
      checkCUDAError("hipMemcpy for idataSwap failed");

      hipFree(idataSwap);
    }

    /**
    * Performs stream compaction on paths, storing the result into odata.
    * All zeroes are discarded.
    *
    * @param n      The number of elements in idata.
    * @param odata  The array into which to store elements.
    * @param idata  The array of elements to compact.
    * @returns      The number of elements remaining after compaction.
    */
    int compactPaths(int n, PathSegment *odata, PathSegment *idata, int* bools_arr, int* indices_arr) {

      dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

      // Copy from CPU to CPU
      hipMemcpy(odata, idata, n * sizeof(PathSegment), hipMemcpyDeviceToDevice);
      checkCUDAError("hipMemcpy for odata failed");


      // Map input array to a temp array of 0s and 1s
      kernMapToBooleanPaths << <fullBlocksPerGrid, blockSize >> >(n, bools_arr, idata);
      checkCUDAError("kernMapToBoolean failed");

      // Scan
      scan(n, indices_arr, bools_arr);

      // Scatter
      kernScatterPaths << <fullBlocksPerGrid, blockSize >> >(n, odata, idata, bools_arr, indices_arr);
      checkCUDAError("kernScatter failed");

      // Copy from CPU to CPU
      hipMemcpy(idata, odata, n * sizeof(PathSegment), hipMemcpyDeviceToDevice);
      checkCUDAError("hipMemcpy for odataSwap failed");

      // Grab remaining number of elements
      int remainingNBools = 0;
      hipMemcpy(&remainingNBools, bools_arr + n - 1, sizeof(PathSegment), hipMemcpyDeviceToHost);

      int remainingNIndices = 0;
      hipMemcpy(&remainingNIndices, indices_arr + n - 1, sizeof(PathSegment), hipMemcpyDeviceToHost);

      return remainingNBools + remainingNIndices;
    }
  }
}
